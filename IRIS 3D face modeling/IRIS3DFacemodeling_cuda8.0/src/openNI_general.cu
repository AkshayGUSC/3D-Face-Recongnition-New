#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

//-----------------------------------------------------------------------------------------
//	Some of this file is part of the "USC IRIS 3D face modeler" 
//      developed at the University of Southern California by
//      Matthias Hernandez, Jongmoo Choi, Gerard Medioni, 
//      Published: Laser Scan Quality 3-D Face Modeling Using a Low-Cost Depth Camera, EUSIPCO 2012.
//-----------------------------------------------------------------------------------------
//      Copyright (c) 2012 University of Southern California.  All Rights Reserved.

#include "stdafx.h"
#include "openNI_general.h"

#include "hip/hip_runtime.h"

#include <hip/hip_runtime_api.h>
#include <helper_cuda_drvapi.h>
#include <helper_cuda_gl.h>
#include <helper_timer.h>
#include <helper_string.h>
#include <hip/hip_vector_types.h>
#include <helper_image.h>
#include <helper_functions.h>
#include <>

#include <hip/device_functions.h>

//////////////////////////////////////////////
// TODO REMOVE THE WEIRD TEXTURE CRAP ????? //
//////////////////////////////////////////////






#ifndef _GENERAL_CUDA_H
#define _GENERAL_CUDA_H


/*
    Because a 2D gaussian mask is symmetry in row and column,
    here only generate a 1D mask, and use the product by row 
    and column index later.

    1D gaussian distribution :
        g(x, d) -- C * exp(-x^2/d^2), C is a constant amplifier

    parameters:
    og - output gaussian array in global memory
    delta - the 2nd parameter 'd' in the above function
    radius - half of the filter size
             (total filter size = 2 * radius + 1)
*/
//use only one block


/********************/
/********************/
/* BILATERAL FILTER */
/********************/
/********************/



__global__ void
d_generate_gaussian(float *og, float delta, int radius)  {
    int x = blockIdx.x * blockDim.x + threadIdx.x;//threadIdx.x;
    og[threadIdx.x] = __expf(-(x * x) /
        (2 * delta * delta));
}                                     

void initGaussian(float *d_gaussian, float gaussian_delta, int fr_g) {
	d_generate_gaussian <<< 1, fr_g>>>(d_gaussian, gaussian_delta, fr_g);
}


/*
    Euclidean Distance (x, y, d) = exp((|x - y| / d)^2 / 2)
*/
__device__ float euclideanLen(float ax, float ay, float bx, float by, float d) {
    float mod = (bx - ax) * (bx - ax) +
                (by - ay) * (by - ay);

    return __expf(-mod / (2 * d * d));
}


/*
    Depth Distance (x, y, d) = exp((|x - y| / d)^2 / 2)
*/
__device__ float euclideanLen(float az, float bz, float d) {
    float mod = (bz - az) * (bz - az);

    return __expf(-mod / (2 * d * d));
}


//extern "C"
void copyImage(uint *h_src, uint *d_dest,
               int width, int height) {
	hipMemcpy(d_dest, h_src, width * height * sizeof(uint), hipMemcpyHostToDevice);
   // cutilSafeCall(hipMemcpy(d_dest, h_src, width * height * sizeof(uint), hipMemcpyHostToDevice));
}

#endif




#ifndef _FILTER_KERNEL_H_
#define _FILTER_KERNEL_H_


/*
    Perform a simple bilateral filter.

    Bilateral filter is a nonlinear filter that is a mixture of range 
    filter and domain filter, the previous one preserves crisp edges and 
    the latter one filters noise. The intensity value at each pixel in 
    an image is replaced by a weighted average of intensity values from 
    nearby pixels.

    The weight factor is calculated by the product of domain filter
    component(using the gaussian distribution as a spatial distance) as 
    well as range filter component(Euclidean distance between center pixel
    and the current neighbor pixel). Because this process is nonlinear, 
    the sample just uses a simple pixel by pixel step. 

    Texture fetches automatically clamp to edge of image. 1D gaussian array
    is mapped to a 1D texture instead of using shared memory, which may 
    cause severe bank conflict.

    Threads are y-pass(column-pass), because the output is coalesced.

    Parameters
    od - pointer to output data in global memory
    d_f - pointer to the 1D gaussian array
    e_d - euclidean delta
    w  - image width
    h  - image height
    r  - filter radius
*/






//column pass using coalesced global memory reads
__global__ void
d_bilateral_filter_and_3D_conversion(float *pReal, float *pDepth, float *gaussian, bool *validity,
                   float s_s, float s_r, int w, int h, int r)
{
    uint x = blockIdx.x * blockDim.x + threadIdx.x;
    uint y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < w && y < h) {
        float sum = 0.0f;
        float factor =0.0f;
	
		int max_i = w*h;
		int i1=y*w+x, 
			i2=i1+max_i, 
			i3=i2+max_i;

		float tz=0.0f;
		float centerz=pDepth[i1];
		float curPixz=0.0f;
		float Z=0.0f;

		//////////////////////////////////////////////////////////////
		// Apply the bilateral filter
		if (validity[i1]) {//(centerz<MAX_DEPTH && centerz>MIN_DEPTH){

			for(int i = -r; i <= r; i++)  {
				for(int j = -r; j <= r; j++)  {

					//curPixx = (x + j);
					//curPixy = (y + i);
					curPixz = pDepth[(y + i)*w + (x + j)];

					factor =  euclideanLen(curPixz, centerz, s_r)	//range factor
							* gaussian[abs(j)]*gaussian[abs(i)]; //euclideanLen(curPixx, curPixy, centerx, centery, s_s); 

					if (validity[i1]) {
						tz += factor * curPixz;
						sum += factor;
					}
				}
			}
			if (sum > 0.0f)
				Z = tz/sum;
		} 

		//////////////////////////////////////////////////////////////
		// Convert to 3D coordinate

		// Set the Z coordinate
		pReal[i3] = Z;

		// Find the X and Y coordinate
		if (Z > MIN_DEPTH && Z < MAX_DEPTH){
			float X_rw = ( (float)x /(float)w -0.5f)*Z*XtoZ;
			float Y_rw = (0.5f-(float)y / (float)h)*Z*YtoZ;

			pReal[i1] = X_rw;
			pReal[i2] = Y_rw;
		} else {
			pReal[i1] = 0.0f;
			pReal[i2] = 0.0f;
			pReal[i3] = 0.0f;
		}
    }
}


//column pass using coalesced global memory reads
__global__ void
d_bilateral_filter(float *pReal, float *pDepth, float *gaussian, bool *validity,
                   float s_s, float s_r, int w, int h, int r)
{
    uint x = blockIdx.x * blockDim.x + threadIdx.x;
    uint y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < w && y < h) {
        float sum = 0.0f;
        float factor =0.0f;
	
		int i1=y*w+x;

		float tz=0.0f;
		float centerz=pDepth[i1];
		float curPixz=0.0f;
		float Z=0.0f;

		//////////////////////////////////////////////////////////////
		// Apply the bilateral filter
		if (validity[i1]) {//(centerz<MAX_DEPTH && centerz>MIN_DEPTH){

			for(int i = -r; i <= r; i++)  {
				for(int j = -r; j <= r; j++)  {

					//curPixx = (x + j);
					//curPixy = (y + i);
					curPixz = pDepth[(y + i)*w + (x + j)];

					factor =  euclideanLen(curPixz, centerz, s_r)	//range factor
							* gaussian[abs(j)]*gaussian[abs(i)]; //euclideanLen(curPixx, curPixy, centerx, centery, s_s); 

					if (validity[i1]) {
						tz += factor * curPixz;
						sum += factor;
					}
				}
			}
			if (sum > 0.0f)
				Z = tz/sum;
		} 

		//////////////////////////////////////////////////////////////
		// Convert to 3D coordinate

		// Set the Z coordinate
		pReal[i1] = Z;
	}
}














/**************************/
/* Compute the PCA matrix */
__global__ void
d_PCA(float *pReal, float *M, int rad, int l, int r, int t, int b, int w, int h, int max_i ) {
	uint x = blockIdx.x * blockDim.x + threadIdx.x;
    uint y = blockIdx.y * blockDim.y + threadIdx.y;
	
	int i0=0;
	int k=0;

	float	xi=0.0f, yi=0.0f, zi=0.0f,
			x_=0.0f, y_=0.0f, z_=0.0f;

	if (x < w && y < h) {
		int max_i2 = 2*max_i;
		int i1=y*w+x; 
		//	i2=i1+max_i, 
		//	i3=i2+max_i;

		float *MM = &M[i1*9];

		if(x>=l && x<=r && y>=t && y<=b && pReal[i1+max_i2]!=0.0f) {
			for(int i = -rad; i <= rad; i++)  {
				for(int j = -rad; j <= rad; j++)  {
					i0 = (y-j)*w+(x-i);

					if(pReal[i0]!=0) {
						xi=pReal[i0];
						yi=pReal[i0+max_i];
						zi=pReal[i0+max_i2];

						x_ += xi;
						y_ += yi;
						z_ += zi;
						k++;

						MM[0] += xi*xi;
						MM[1] += xi*yi;
						MM[2] += xi*zi;
						MM[4] += yi*yi;
						MM[5] += yi*zi;
						MM[8] += zi*zi;
					}
				}
			}

			if (k>0) {
				x_ /= k;
				y_ /= k;
				z_ /= k;

				MM[0] = MM[0]/k - x_*x_;
				MM[1] = MM[1]/k - x_*y_;
				MM[2] = MM[2]/k - x_*z_;
				MM[4] = MM[4]/k - y_*y_;
				MM[5] = MM[5]/k - y_*z_;
				MM[8] = MM[8]/k - z_*z_;
				MM[3] = MM[1];
				MM[6] = MM[2];
				MM[7] = MM[5];
			} else {
				for (int i=0; i<9; i++)
					MM[i] = 0.0f;
			}
		} else
			for (int i=0; i<9; i++)
				MM[i] = 0.0f;
	}
}
/**************************************************/



//Perform a matrix decomposition of VV: 
// After running, VV will store the eigenvecotrs and dd the eigenvalues
// CODE FROM http://openslam.informatik.uni-freiburg.de/data/svn/gmapping/trunk/scanmatcher/eig3.cpp , copied from the public domain Java Matrix library JAMA.
__global__ void
d_eigendecomposition(float *VV, float *dd, int w, int h ) {
	//float	xi=0.0f, yi=0.0f, zi=0.0f,
	//		x_=0.0f, y_=0.0f, z_=0.0f;
	int n=3;
	
	//int L=n*(n-1);
	float e[3];

	uint x = blockIdx.x * blockDim.x + threadIdx.x;
    uint y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < w && y < h) {
		int i1=y*w+x; 

		float *d = &dd[i1*3];
		float *V = &VV[i1*9];

		if (	V[0]!=0.0f || V[1]!=0.0f || V[2]!=0.0f ||
				V[3]!=0.0f || V[4]!=0.0f || V[5]!=0.0f ||
				V[6]!=0.0f || V[7]!=0.0f || V[8]!=0.0f	)  {

			for (int j = 0; j < 3; j++) {
				d[j] = V[6+j];
			}

			// Householder reduction to tridiagonal form.

			for (int i = 2; i > 0; i--) {

			// Scale to avoid under/overflow.

				float scale = 0.0f;
				float h = 0.0f;
				for (int k = 0; k < i; k++) {
					scale = scale + fabs(d[k]);
				}
				if (scale == 0.0) {
					e[i] = d[i-1];
					for (int j = 0; j < i; j++) {
						d[j] = V[(i-1)*n+j];
						V[i*n+j] = 0.0;
						V[j*n+i] = 0.0;
					}
				} else {

			  // Generate Householder vector.

					for (int k = 0; k < i; k++) {
						d[k] /= scale;
						h += d[k] * d[k];
					}
					float f = d[i-1];
					float g = sqrt(h);
					if (f > 0) {
						g = -g;
					}
					e[i] = scale * g;
					h = h - f * g;
					d[i-1] = f - g;
					for (int j = 0; j < i; j++) {
						e[j] = 0.0;
					}

				  // Apply similarity transformation to remaining columns.

					for (int j = 0; j < i; j++) {
						f = d[j];
						V[j*n+i] = f;
						g = e[j] + V[j*n+j] * f;
						for (int k = j+1; k <= i-1; k++) {
							g += V[k*n+j] * d[k];
							e[k] += V[k*n+j] * f;
						}
						e[j] = g;
					}
					f = 0.0;
					for (int j = 0; j < i; j++) {
						e[j] /= h;
						f += e[j] * d[j];
					}
					float hh = f / (h + h);
					for (int j = 0; j < i; j++) {
						e[j] -= hh * d[j];
					}
					for (int j = 0; j < i; j++) {
						f = d[j];
						g = e[j];
						for (int k = j; k <= i-1; k++) {
							V[k*n+j] -= (f * e[k] + g * d[k]);
						}
						d[j] = V[(i-1)*n+j];
						V[i*n+j] = 0.0;
					}
				}
				d[i] = h;
			}

			// Accumulate transformations.

			for (int i = 0; i < n-1; i++) {
				V[(n-1)*n+i] = V[i*n+i];
				V[i*n+i] = 1.0;
				float h = d[i+1];
				if (h != 0.0) {
					for (int k = 0; k <= i; k++) {
						d[k] = V[k*n+i+1] / h;
					}
					for (int j = 0; j <= i; j++) {
						float g = 0.0;
						for (int k = 0; k <= i; k++) {
						  g += V[k*n+i+1] * V[k*n+j];
						}
						for (int k = 0; k <= i; k++) {
						  V[k*n+j] -= g * d[k];
						}
					}
				}
				for (int k = 0; k <= i; k++) {
					V[k*n+i+1] = 0.0;
				}
			}
			for (int j = 0; j < n; j++) {
				d[j] = V[(n-1)*n+j];
				V[(n-1)*n+j] = 0.0;
			}
			V[(n-1)*n+n-1] = 1.0;
			e[0] = 0.0;

			
			for (int i = 1; i < n; i++) {
				e[i-1] = e[i];
			}
			e[n-1] = 0.0;

			float f = 0.0;
			float tst1 = 0.0;
			float eps = pow(2.0,-52.0);
			for (int l = 0; l < 3; l++) {

				// Find small subdiagonal element

				tst1 = MAX(tst1,fabs(d[l]) + fabs(e[l]));
				int m = l;
				while (m < 3) {
					if (fabs(e[m]) <= eps*tst1) {
						break;
					}
					m++;
				}

				// If m == l, d[l] is an eigenvalue,
				// otherwise, iterate.

				if (m > l) {
					int iter = 0;
					do {
						iter = iter + 1;  // (Could check iteration count here.)

						// Compute implicit shift

						float g = d[l];
						float p = (d[l+1] - g) / (2.0 * e[l]);
						float r = sqrt(p*p+1.0f);
						if (p < 0) {
							r = -r;
						}
						d[l] = e[l] / (p + r);
						d[l+1] = e[l] * (p + r);
						float dl1 = d[l+1];
						float h = g - d[l];
						for (int i = l+2; i < n; i++) {
							d[i] -= h;
						}
						f = f + h;

						// Implicit QL transformation.

						p = d[m];
						float c = 1.0;
						float c2 = c;
						float c3 = c;
						float el1 = e[l+1];
						float s = 0.0;
						float s2 = 0.0;
						for (int i = m-1; i >= l; i--) {
							c3 = c2;
							c2 = c;
							s2 = s;
							g = c * e[i];
							h = c * p;
							r = sqrt(p*p+e[i]*e[i]);
							e[i+1] = s * r;
							s = e[i] / r;
							c = p / r;
							p = c * d[i] - s * g;
							d[i+1] = h + s * (c * g + s * d[i]);

							// Accumulate transformation.

							for (int k = 0; k < n; k++) {
								h = V[k*n+i+1];
								V[k*n+i+1] = s * V[k*n+i] + c * h;
								V[k*n+i] = c * V[k*n+i] - s * h;
							}
						}
						p = -s * s2 * c3 * el1 * e[l] / dl1;
						e[l] = s * p;
						d[l] = c * p;

							// Check for convergence.

					} while (fabs(e[l]) > eps*tst1);
				}
				d[l] = d[l] + f;
				e[l] = 0.0;
			}
				// Sort eigenvalues and corresponding vectors.

			for (int i = 0; i < n-1; i++) {
				int k = i;
				float p = d[i];
				for (int j = i+1; j < n; j++) {
					if (d[j] < p) {
						k = j;
						p = d[j];
					}
				}
				if (k != i) {
					d[k] = d[i];
					d[i] = p;
					for (int j = 0; j < n; j++) {
						p = V[j*n+i];
						V[j*n+i] = V[j*n+k];
						V[j*n+k] = p;
					}
				}
			}
		}
	}

}
/**************************************************/


//Save the normals
__global__ void
d_Convert2Normals(float *V, float *norm, int w, int h, int max_i ) {
	uint x = blockIdx.x * blockDim.x + threadIdx.x;
    uint y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < w && y < h) {
		int i = y*w+x;
		int i9 = 9*i;

		if ( V[i9+6]<0.0f) {
			norm[i]			= -V[i9];
			norm[i+max_i]	= -V[i9+3];
			norm[i+2*max_i]	= -V[i9+6];
		} else {
			norm[i]			= V[i9];
			norm[i+max_i]	= V[i9+3];
			norm[i+2*max_i]	= V[i9+6];
		}
	}
}
/**************************************************/



//Save the normals
__global__ void
d_Convert2Normals(float *V, float *norm, char *nData, int w, int h, int max_i ) {
	uint x = blockIdx.x * blockDim.x + threadIdx.x;
    uint y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < w && y < h) {
		int i = y*w+x;
		int i9 = 9*i;

		if ( V[i9+6]<0.0f) {
			norm[i]			= -V[i9];
			norm[i+max_i]	= -V[i9+3];
			norm[i+2*max_i]	= -V[i9+6];
		} else {
			norm[i]			= V[i9];
			norm[i+max_i]	= V[i9+3];
			norm[i+2*max_i]	= V[i9+6];
		}

		nData[i] = (char) (norm[MAX_I2+i]*255.0f+0.5f);
	}
}
/**************************************************/







//column pass using coalesced global memory reads
__global__ void
d_compute_normals(float *out, float *input,
				  int l, int r, int t, int b,
                  int w, int h, int max_i) {
    uint x = /*__umul24(blockIdx.x, blockDim.x)*/ blockIdx.x * blockDim.x + threadIdx.x;
    uint y = /*__umul24(blockIdx.y, blockDim.y)*/ blockIdx.y * blockDim.y + threadIdx.y;

	float	Ux=0.0f,  Uy=0.0f,  Uz=0.0f,
			Vx=0.0f,  Vy=0.0f,  Vz=0.0f,
			UVx=0.0f, UVy=0.0f, UVz=0.0f,
			n2=0.0f;
	//int MIN_DEPTH=450, MAX_DEPTH=800;

    if (x < w && y < h) {

		int i1x = y*w+x,	i1y = i1x+max_i,	i1z = i1y+max_i;
		out[i1x]= 0.0f;
		out[i1y]= 0.0f;
		out[i1z]= 0.0f;

		if ( x>=l && x<=r && y>=t && y<=b ) {

			int i2x = i1x+w,	i2y = i2x+max_i,	i2z = i2y+max_i;
			int i3x = i1x+1,	i3y = i1y+1,		i3z = i1z+1;
			

			if ( input[i1z]>=MIN_DEPTH && input[i1z]<=MAX_DEPTH) {

				if ( (x+1)<w && (y+1)<h) {	

					if (input[i1z]!=0.0f && input[i2z]!=0.0f && input[i3z]!=0.0f) {
						Ux = input[i2x]-input[i1x];
						Uy = input[i2y]-input[i1y];
						Uz = input[i2z]-input[i1z];

						Vx = input[i3x]-input[i1x];
						Vy = input[i3y]-input[i1y];
						Vz = input[i3z]-input[i1z];

						UVx	= Uy*Vz-Vy*Uz;
						UVy	= Vx*Uz-Ux*Vz;
						UVz	= Ux*Vy-Vx*Uy;

						n2 = UVx*UVx + UVy*UVy + UVz*UVz;

						if (n2>0.0f) {
							n2 = pow(n2, 0.5f);

							out[i1x]= UVx/n2;
							out[i1y]= UVy/n2;
							out[i1z]= UVz/n2;
						} 
					} 
				} 
			}  
		}
    }
}
/**************************************************/


__global__ void 
d_convertP2RW(float *pDepth, float *pReal, int w, int h, int max_i){

	uint x = /*__umul24(blockIdx.x, blockDim.x)*/ blockIdx.x * blockDim.x + threadIdx.x;
    uint y = /*__umul24(blockIdx.y, blockDim.y)*/ blockIdx.y * blockDim.y + threadIdx.y;

	//int MAX_DEPTH = 800;
	//int MIN_DEPTH = 400;

	//float FovH = 1.0144686707507438f;
    //float FovV =0.78980943449644714f;

	int i1 = (y * w + x),
		i2 = i1 + max_i,
		i3 = i2 + max_i;

	float Z = pDepth[i1];

    if (Z > MIN_DEPTH && Z < MAX_DEPTH){
        float X_rw = ( (float)x /(float)w -0.5f)*Z*XtoZ;
        float Y_rw = (0.5f-(float)y / (float)h)*Z*YtoZ;

		pReal[i1] = X_rw;
		pReal[i2] = Y_rw;
		pReal[i3] = Z;
	} else {
		pReal[i1] = 0.0f;
		pReal[i2] = 0.0f;
		pReal[i3] = 0.0f;
	}
}
/**************************************************/




//column pass using coalesced global memory reads
__global__ void d_display_images(	char *out_rgb, char *input_rgb,
									char *out_d, float *input_d,
									float max_d, float min_d, float normalize, int *palette,
									int w, int h) {
    uint x = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    uint y = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;

	int i_d = y*w+x,
		i_r = 3*i_d,
		i_g = i_r+1,
		i_b = i_g+1;

	out_rgb[i_r] = input_rgb[i_b];
	out_rgb[i_g] = input_rgb[i_g];
	out_rgb[i_b] = input_rgb[i_r];

	if (input_d[i_d] < min_d || input_d[i_d] > max_d) {
		int value = (int)(255.5f -(input_d[i_d]-(float)min_d)/(float)normalize);//palette

		out_d[i_r] = palette[value];
		out_d[i_g] = palette[value+256];
		out_d[i_b] = palette[value+512];
	} else {
		out_d[i_r] = 0;
		out_d[i_g] = 0;
		out_d[i_b] = 0;
	}
}
/**************************************************/



/***********************************************/
/* Calculate the norm of the vectorial product */
__global__ void d_cross_p(	float *mReal, float *fReal, float *sin_pf, int *index_f,
							int nbm, int nbf) {
    uint x = threadIdx.x;
	
	float Xp = fReal[x];
	float Yp = fReal[x+nbf];
	float Zp = fReal[x+2*nbf];

	float Xf = mReal[index_f[x]];
	float Yf = mReal[index_f[x]+nbm];
	float Zf = mReal[index_f[x]+2*nbm];

	float Xc = Yp*Zf - Yf*Zp;
	float Yc = Zp*Xf - Zf*Xp;
	float Zc = Xp*Yf - Xf*Yp;

	sin_pf[x] = Xc*Xc + Yc*Yc+ Zc*Zc;
	
}
/**************************************************/



//// Compute the normals
//void display_images(char *h_rgb_img, char *h_d_img,
//					char *h_rgb, float *h_d,
//					float max_d, float min_d, float normalize, int *palette,
//                    int width, int height) {
//	int max_i=width * height,
//		max_i3=max_i*3;
//
//	copyHostToCUDA_char(rgb, max_i3);
//	copyHostToCUDA(d, max_i);
//
//	dim3 gridSize((width + 16 - 1) / 16, (height + 16 - 1) / 16);
//    dim3 blockSize(16, 16);
//
//	
//	d_display_images<<< gridSize, blockSize>>>(d_rgb_img, d_rgb, d_d_img, d_d, max_d, min_d, normalize, palette, width, height);
//	
//	copyCUDAToHost_char(rgb_img, max_i3);
//	//copyCUDAToHost_char(d_img, max_i3);
//}

















/*
    Perform 2D bilateral filter on image using CUDA

    Parameters:
    d_dest - pointer to destination image in device memory
    width  - image width
    height - image height
    e_d    - euclidean delta
    radius - filter radius
    iterations - number of iterations
*/





/************************************/
/* Apply bilateral filter on h_orig */
/* Output is converted to 3D in pReal */
void bilateralFilterRGBA(float *h_pDepth, 
						 float *d_pDepth, 
						 float *h_pReal, 
						 float *d_pReal, 
						 float *d_gaussian,
						 bool *d_validity,
                         int width, int height,
                         float s_s, float s_r, int radius, int iterations,
                         int nthreads) {
	int max_i=width * height;
	
	dim3 gridSize((width + 16 - 1) / 16, (height + 16 - 1) / 16);
    dim3 blockSize(16, 16);

	copyHostToCUDA(pDepth, max_i);	

    for(int i=0; i<iterations; i++)  {
        // sync host and start kernel computation timer
        //synchronizeCUDA();
        d_bilateral_filter_and_3D_conversion<<< gridSize, blockSize>>>(d_pReal, d_pDepth, d_gaussian, d_validity, s_s, s_r, width, height, radius);
    }

	copyCUDAToHost(pReal,3*max_i);	
}
/*******************************************/



/************************************/
/* Apply bilateral filter on h_orig */
/* Output is converted to 3D in pReal */
void bilateralFilter(	 float *h_pDepth, 
						 float *d_pDepth, 
						 float *h_pReal, 
						 float *d_pReal, 
						 float *d_gaussian,
						 bool *d_validity,
                         int width, int height,
                         float s_s, float s_r, int radius, int iterations,
                         int nthreads) {
	int max_i=width * height;
	
	dim3 gridSize((width + 16 - 1) / 16, (height + 16 - 1) / 16);
    dim3 blockSize(16, 16);

	copyHostToCUDA(pDepth, max_i);	

    for(int i=0; i<(iterations/2); i++)  {
        // sync host and start kernel computation timer
        //synchronizeCUDA();
        d_bilateral_filter<<< gridSize, blockSize>>>(d_pReal, d_pDepth, d_gaussian, d_validity, s_s, s_r, width, height, radius);
		d_bilateral_filter<<< gridSize, blockSize>>>(d_pDepth, d_pReal, d_gaussian, d_validity, s_s, s_r, width, height, radius);
    }

	copyCUDAToHost(pDepth,max_i);	
}
/*******************************************/






/*******************************************/
/* Convert projective to Real World on GPU */
void convertP2RW(	float *d_pDepth, float *h_pDepth,
					float *d_pReal, float *h_pReal, 
					int width, int height) {
	int max_i=width * height;
	
	dim3 gridSize((width + 16 - 1) / 16, (height + 16 - 1) / 16);
    dim3 blockSize(16, 16);

	//copyHostToCUDA(pDepth, max_i);	

	//synchronizeCUDA();
    d_convertP2RW<<< gridSize, blockSize>>>(d_pDepth, d_pReal, width, height, max_i); // BUG UNTIL 159- WORKS AT 160

	copyCUDAToHost(pReal,3*max_i);	
	//int MIN_DEPTH=400,MAX_DEPTH=800;
	int x=0, y=0;

	int i2 = 0,
		i3 = 0;
	int i_max = 160*width;

	for (int i=0; i<i_max; i++) {
		x=i%width;
		y=(i-x)/width;
		float Z = h_pDepth[i];
		i2=i+max_i;
		i3=i2+max_i;
		if (Z > MIN_DEPTH && Z < MAX_DEPTH){
			float X_rw = ( (float)x /(float)width -0.5f)*Z*XtoZ;
			float Y_rw = (0.5f-(float)y / (float)height)*Z*YtoZ;

			h_pReal[i]  = X_rw;
			h_pReal[i2] = Y_rw;
			h_pReal[i3] = Z;
		} else {
			h_pReal[i] = 0.0f;
			h_pReal[i2] = 0.0f;
			h_pReal[i3] = 0.0f;
		}
	}
}
/*******************************************/


/***********************/
/* Compute the normals */
void normalComputation(float *h_pReal, float *h_normals, float* d_pReal, float *d_normals, FaceBox maskBox,
                           int width, int height) {
	int max_i=width * height,
		max_i3=max_i*3;
	dim3 gridSize((width + 16 - 1) / 16, (height + 16 - 1) / 16);
	dim3 blockSize(16, 16);


	//copyHostToCUDA(pReal, max_i3);	

	// sync host and start kernel computation timer
	//synchronizeCUDA();
	d_compute_normals<<< gridSize, blockSize>>>(d_normals, d_pReal, maskBox.getLeftX(), maskBox.getRightX(), maskBox.getTopY(), maskBox.getBottomY(), width, height, max_i); 

	copyCUDAToHost(normals,max_i3);	
}
/*******************************************/


/**************************************************/
/* Apply the PCA algorithm to compute the normals */
void PCA(float* d_pReal, 
		 float *d_M, float *d_d,
		 float *d_normals, float *h_normals,
		 int l, int r, int t, int b,
		 int radius,
		 int width, int height) {
	// TODO
	// MallocCUDA M[9*MAX_I]
	// mallocCUDA d[3*MAX_I]
	int max_i  = width * height,
		max_i3 = max_i*3;
	dim3 gridSize((width + 16 - 1) / 16, (height + 16 - 1) / 16);
	dim3 blockSize(16, 16);	

	// Compute the PCA matrices
	//synchronizeCUDA();
	d_PCA<<< gridSize, blockSize>>>(d_pReal, d_M, radius, l, r, t, b, width, height, max_i ); 
	// Perform the eigenvalue decomposition
	//synchronizeCUDA();
	d_eigendecomposition<<< gridSize, blockSize>>>(d_M, d_d, width, height );
	// Extract the normals
	//synchronizeCUDA();
	d_Convert2Normals<<< gridSize, blockSize>>>(d_M, d_normals, width, height, max_i );

	copyCUDAToHost(normals,max_i3);	
}
/*******************************************/








/**************************************************/
/* Apply the PCA algorithm to compute the normals */
void PCA(float* d_pReal, 
		 float *d_M, float *d_d,
		 float *d_normals, float *h_normals,
		 char *d_nData, char *h_nData,
		 int l, int r, int t, int b,
		 int radius,
		 int width, int height) {
	// TODO
	// MallocCUDA M[9*MAX_I]
	// mallocCUDA d[3*MAX_I]
	int max_i  = width * height,
		max_i3 = max_i*3;
	dim3 gridSize((width + 16 - 1) / 16, (height + 16 - 1) / 16);
	dim3 blockSize(16, 16);

	// Compute the PCA matrices
	//synchronizeCUDA();
	d_PCA<<< gridSize, blockSize>>>(d_pReal, d_M, radius, l, r, t, b, width, height, max_i ); 
	// Perform the eigenvalue decomposition
	//synchronizeCUDA();
	d_eigendecomposition<<< gridSize, blockSize>>>(d_M, d_d, width, height );
	// Extract the normals
	//synchronizeCUDA();
	d_Convert2Normals<<< gridSize, blockSize>>>(d_M, d_normals, d_nData, width, height, max_i );

	copyCUDAToHost(normals,max_i3);	
	copyCUDAToHost_char(nData,max_i);	
}
/*******************************************/





/***************************************************************/
/* Calculate the norm of the vectorial product for every point */
void cross_p(float *d_mNormals, float *h_mNormals, int nbm,
			 float *d_fNormals, float *h_fNormals, int nbf,
			 float *d_cross, float *h_cross,
			 int *d_index_f, int *h_index_f) {
	
	copyHostToCUDA(mNormals,nbm*3);
	copyHostToCUDA(fNormals,nbf*3);
	copyHostToCUDA_int(index_f,nbf);


	// Compute the Norm of the cross products
	//synchronizeCUDA();
	d_cross_p<<< nbf, 1 >>>(d_mNormals, d_fNormals, d_cross, d_index_f, nbm, nbf ); 

	copyCUDAToHost(cross,nbf);	
}
/*******************************************/







/*********************/
/* Display one pixel */
__global__ void  
d_displayDepthImage	(	float *pReal,
						char *data,
						bool *validity,
						int *palette, bool isHeat,
						int width, int height) {
	uint x = blockIdx.x * blockDim.x + threadIdx.x;
    uint y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < width && y < height) {
		int i=y*width+x,
			i3=3*i; 
		
		if (validity[i]) {
			int value = (int)(255-(float)(pReal[MAX_I2+i]-MIN_DEPTH)/NORMALIZE_DISPLAY);
			if (!isHeat) {
				data[i3]	= value;
				data[i3+1]	= data[i3];
				data[i3+2]	= data[i3];
			} else {
				data[i3]	= palette[value];
				data[i3+1]	= palette[value+SIZE_CHANNEL_PALETTE];
				data[i3+2]	= palette[value+SIZE_CHANNEL_PALETTE_2];
			}
		} else {
			data[i3]	= 0;
			data[i3+1]	= data[i3];
			data[i3+2]	= data[i3];
		}		
	}
}
/*******************************************/


/*************************/
/* Display the depth map */
void displayDepthImage	(	float *d_pReal,
							char *d_data, char *h_data, 
							bool  *d_validity,
							int *d_palette, bool isHeat,
							int width, int height) {
	dim3 gridSize((width + 16 - 1) / 16, (height + 16 - 1) / 16);
	dim3 blockSize(16, 16);
	
	d_displayDepthImage<<< gridSize, blockSize>>>(d_pReal, d_data, d_validity, d_palette, isHeat, width, height ); 			
}
/*******************************************/














__global__ 
void d_render_monkey(	float *h_monkey_1, float *h_monkey_2, char *data,
						int nb,
						float *h_R, float *h_t,
						float scale) {
	int i_X = blockIdx.x * blockDim.x + threadIdx.x;
	int i_Y = i_X + nb,
		i_Z = i_Y + nb;

	// Apply R,t
	h_monkey_2[i_X] = h_R[0]*h_monkey_1[i_X] + h_R[1]*h_monkey_1[i_Y] + h_R[2]*h_monkey_1[i_Z];
	h_monkey_2[i_Y] = h_R[3]*h_monkey_1[i_X] + h_R[4]*h_monkey_1[i_Y] + h_R[5]*h_monkey_1[i_Z];
	h_monkey_2[i_Z] = h_R[6]*h_monkey_1[i_X] + h_R[7]*h_monkey_1[i_Y] + h_R[8]*h_monkey_1[i_Z] + 5.0f;
	
	// Project
	h_monkey_2[i_X] = (h_monkey_2[i_X]/h_monkey_2[i_Z]/XtoZ + 0.5f)*(float)XN_HR_X_RES;
	h_monkey_2[i_Y] = (-h_monkey_2[i_Y]/h_monkey_2[i_Z]/YtoZ + 0.5f)*(float)XN_HR_Y_RES;

	// Move and scale
	h_monkey_2[i_X] = ( ((h_monkey_2[i_X]-290.0f)/4.0f+30.0f)*scale + 0.5f);
	h_monkey_2[i_Y] = ( ((h_monkey_2[i_Y]-245.0f)/4.0f+30.0f)*scale + 0.5f);
		
	int x = (int)h_monkey_2[i_X];
	int y = (int)h_monkey_2[i_Y];

	int ind = y*XN_VGA_X_RES*3+x*3;
	if (ind < 921597)
	{
	data[ind]	= (char)255;
	data[ind+1] = (char)255;
	data[ind+2] = (char)255;
	}
}

void render_monkey(		float *h_monkey_1, float *d_monkey_1, 
						float *d_monkey_2, 
						char *h_dData, char *d_dData,
						int nb,
						float *d_R, float *d_t,
						float scale, bool isDepth){
	int nb_g = (nb + BLOCK_SIZE-1) / BLOCK_SIZE;

	if (!isDepth)
		copyHostToCUDA_char(dData, MAX_I3);

	// sync host and start kernel computation timer
	//synchronizeCUDA();
	//d_render_monkey<<< nb_g, BLOCK_SIZE >>>(d_monkey_1, d_monkey_2, d_dData, nb, d_R, d_t, scale); 	
}


#endif
