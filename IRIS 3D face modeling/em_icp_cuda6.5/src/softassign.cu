#include "hip/hip_runtime.h"
/*
  Copyright (c) 2010 Toru Tamaki

  Permission is hereby granted, free of charge, to any person
  obtaining a copy of this software and associated documentation
  files (the "Software"), to deal in the Software without
  restriction, including without limitation the rights to use,
  copy, modify, merge, publish, distribute, sublicense, and/or sell
  copies of the Software, and to permit persons to whom the
  Software is furnished to do so, subject to the following
  conditions:

  The above copyright notice and this permission notice shall be
  included in all copies or substantial portions of the Software.

  THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,
  EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES
  OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND
  NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT
  HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY,
  WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
  FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR
  OTHER DEALINGS IN THE SOFTWARE.
*/


#include <iostream>
#include <cstdio>

#include <cutil.h>
#include <hipblas.h>

// uncomment if you do not use the viewer.
//#define NOVIEWER

#include "3dregistration.h"
#include "engine.h"

using namespace std;




__global__ static void
updateM(int rowsM, int colsM, int pitchM,
	float* d_Xx, float* d_Xy, float* d_Xz, 
	float* d_Yx, float* d_Yy, float* d_Yz,
	float* d_R, float* d_t,
	float* d_M,
	float T_cur, float alpha){
  

  int r =  blockIdx.x * blockDim.x + threadIdx.x;
  int c =  blockIdx.y * blockDim.y + threadIdx.y;

  // Shared memory
  __shared__ float XxShare[BLOCK_SIZE];
  __shared__ float XyShare[BLOCK_SIZE];
  __shared__ float XzShare[BLOCK_SIZE];
  __shared__ float YxShare[BLOCK_SIZE];
  __shared__ float YyShare[BLOCK_SIZE];
  __shared__ float YzShare[BLOCK_SIZE];
  __shared__ float RShare[9]; // BLOCK_SIZE >= 9 is assumed
  __shared__ float tShare[3]; // BLOCK_SIZE >= 3 is assumed
  
  if(threadIdx.y == 0)
    if(threadIdx.x < 9){
      RShare[threadIdx.x] = d_R[threadIdx.x];
      if(threadIdx.x < 3)
	tShare[threadIdx.x] = d_t[threadIdx.x];
    }
  
  if(r < rowsM && c < colsM){ // check for only inside the matrix M
    
    if(threadIdx.y == 0){
      XxShare[threadIdx.x] = d_Xx[r];
      XyShare[threadIdx.x] = d_Xy[r];
      XzShare[threadIdx.x] = d_Xz[r];
    }
    if(threadIdx.x == 0){
      YxShare[threadIdx.y] = d_Yx[c];
      YyShare[threadIdx.y] = d_Yy[c];
      YzShare[threadIdx.y] = d_Yz[c];
    }

    __syncthreads();

#define Xx XxShare[threadIdx.x]
#define Xy XyShare[threadIdx.x]
#define Xz XzShare[threadIdx.x]
#define Yx YxShare[threadIdx.y]
#define Yy YyShare[threadIdx.y]
#define Yz YzShare[threadIdx.y]
#define R(i) RShare[i]
#define t(i) tShare[i]

// #define Euclid(a,b,c) ((a)*(a)+(b)*(b)+(c)*(c))
//     float tmp =
//       Euclid(Xx - (R(0)*Yx + R(1)*Yy + R(2)*Yz + t(0)),
//      	     Xy - (R(3)*Yx + R(4)*Yy + R(5)*Yz + t(1)),
//      	     Xz - (R(6)*Yx + R(7)*Yy + R(8)*Yz + t(2)) ) - alpha;
    
//     tmp = expf(-tmp/T_cur) / sqrtf(T_cur);

     float tmpX = Xx - (R(0)*Yx + R(1)*Yy + R(2)*Yz + t(0));
     float tmpY = Xy - (R(3)*Yx + R(4)*Yy + R(5)*Yz + t(1));
     float tmpZ = Xz - (R(6)*Yx + R(7)*Yy + R(8)*Yz + t(2));

#undef Xx
#undef Xy
#undef Xz
#undef Yx
#undef Yy
#undef Yz
#undef R
#undef t

    __syncthreads();

     tmpX *= tmpX;
     tmpY *= tmpY;
     tmpZ *= tmpZ;

     tmpX += tmpY;
     tmpX += tmpZ;
     tmpX -= alpha;

     tmpX /= T_cur;
     tmpX = expf(-tmpX);
     tmpX /= sqrtf(T_cur);



     d_M[c * pitchM + r] = tmpX;
  }


}

__global__ static void
normalizeMbySinkhorn_row(int rowsM, int colsM, int pitchM,
			 float *d_M,
			 const float *d_sumOfRow,
			 float *d_m_outliers_row
			 ){

  int r =  blockIdx.x * blockDim.x + threadIdx.x;
  int c =  blockIdx.y * blockDim.y + threadIdx.y;

  // Shared memory
  __shared__ float sumOfRowShare[BLOCK_SIZE];


  if(r < rowsM && c < colsM){ // check for only inside the matrix M

    if(threadIdx.y == 0)
      sumOfRowShare[threadIdx.x] = d_sumOfRow[r];

    __syncthreads();

    d_M[c * pitchM + r] /= sumOfRowShare[threadIdx.x];

    if(c == 0) d_m_outliers_row[r] /= sumOfRowShare[threadIdx.x];

    __syncthreads();

  }

}


__global__ static void
normalizeMbySinkhorn_col(int rowsM, int colsM, int pitchM,
			 float *d_M,
			 const float *d_sumOfCol,
			 float *d_m_outliers_col
			 ){

  int r =  blockIdx.x * blockDim.x + threadIdx.x;
  int c =  blockIdx.y * blockDim.y + threadIdx.y;

  // Shared memory
  __shared__ float sumOfColShare[BLOCK_SIZE];

  if(r < rowsM && c < colsM){ // check for only inside the matrix M

    if(threadIdx.x == 0)
      sumOfColShare[threadIdx.y] = d_sumOfCol[c];

    __syncthreads();

    d_M[c * pitchM + r] /= sumOfColShare[threadIdx.y];
    
    if(r == 0) d_m_outliers_col[c] /= sumOfColShare[threadIdx.y];

    __syncthreads();

  }

}



__global__ static void
elementwiseMultiplicationCopy(int rowsM,
			      const float* d_Xx, const float* d_Xy, const float* d_Xz,
			      const float* d_sumOfMRow,
			      float* d_Xx_result, float* d_Xy_result, float* d_Xz_result){

  int r =  blockIdx.x * blockDim.x + threadIdx.x;

  float l_sumOfRow = d_sumOfMRow[r];

  if(r < rowsM){ // check for only inside the matrix M
    d_Xx_result[r] = l_sumOfRow * d_Xx[r];
    d_Xy_result[r] = l_sumOfRow * d_Xy[r];
    d_Xz_result[r] = l_sumOfRow * d_Xz[r];
  }
}



__global__ static void
centeringXorY(int rowsM,
	      const float* d_Xc, float sum,
	      float* d_Xx_result, float* d_Xy_result, float* d_Xz_result){

  // can be work for both row and column
  
  int r =  blockIdx.x * blockDim.x + threadIdx.x;

  // Shared memory
  __shared__ float Xc[3];

  if(threadIdx.x < 3) Xc[threadIdx.x] = d_Xc[threadIdx.x];


  if(r < rowsM){ // check for only inside the matrix M

    __syncthreads();

    d_Xx_result[r] -= Xc[0];
    d_Xy_result[r] -= Xc[1];
    d_Xz_result[r] -= Xc[2];

    __syncthreads();

  }
}





void softassign(const int Xsize, const int Ysize,
		const float* h_X,
		const float* h_Y,
		float* h_R, float* h_t, 
		registrationParameters param
		){

  //
  // initialize paramters
  //

  int JMAX = param.JMAX;
  int I0 = param.I0;
  int I1 = param.I1;
  float T_cur = param.T_0; // current temprature
  float alpha = param.alpha;
  float TFACTR = param.TFACTOR;
  float moutlier = param.moutlier;



  //
  // initialize CUDA
  //
  CUT_DEVICE_INIT(param.argc, param.argv);



  //
  // memory allocation
  //


  // example: memCUDA(Xx, Xsize);   // declare d_Xx. no copy.
#define memCUDA(var,num)						\
  float* d_ ## var; CUDA_SAFE_CALL(hipMalloc((void**) &(d_ ## var), sizeof(float)*num));


  // example:   memHostToCUDA(Xx, Xsize);   // declera d_Xx, then copy h_Xx to d_Xx.
#define memHostToCUDA(var,num)						\
  float* d_ ## var; CUDA_SAFE_CALL(hipMalloc((void**) &(d_ ## var), sizeof(float)*num)); \
  CUDA_SAFE_CALL(hipMemcpy(d_ ## var, h_ ## var, sizeof(float)*num, hipMemcpyHostToDevice));



  memHostToCUDA(X, Xsize*3);
  float* d_Xx = &d_X[Xsize*0];
  float* d_Xy = &d_X[Xsize*1];
  float* d_Xz = &d_X[Xsize*2];

  memHostToCUDA(Y, Ysize*3);
  float* d_Yx = &d_Y[Ysize*0];
  float* d_Yy = &d_Y[Ysize*1];
  float* d_Yz = &d_Y[Ysize*2];

  memCUDA(X_result, Xsize*3);
  float *d_Xx_result = &d_X_result[Xsize*0];
  float *d_Xy_result = &d_X_result[Xsize*1];
  float *d_Xz_result = &d_X_result[Xsize*2];

  memCUDA(Y_result, Ysize*3);
  float *d_Yx_result = &d_Y_result[Ysize*0];
  float *d_Yy_result = &d_Y_result[Ysize*1];
  float *d_Yz_result = &d_Y_result[Ysize*2];

  // center of X, Y
  float h_Xc[3], h_Yc[3];
  memCUDA(Xc, 3);
  memCUDA(Yc, 3);

  // R, t
  memHostToCUDA(R, 3*3);
  memHostToCUDA(t, 3);

  // S for finding R, t
  float h_S[9];
  memCUDA(S, 9);


  // NOTE on matrix M
  // number of rows:     Xsize, or rowsM
  // number of columns : Ysize, or colsM
  // 
  //                    [0th in Y] [1st]  ... [(Ysize-1)] 
  // [0th point in X] [ M(0,0)     M(0,1) ... M(0,Ysize-1)      ] 
  // [1st           ] [ M(1,0)     M(1,1) ...                   ]
  // ...              [ ...                                     ]
  // [(Xsize-1)     ] [ M(Xsize-1, 0)     ... M(Xsize-1,Ysize-1)]
  //
  // 
  // CAUTION on matrix M
  // M is allcoated as a column-maijor format for the use of cublas.
  // This means that you must acces an element at row r and column c as:
  // M(r,c) = M[c * pitchM + r]

  int rowsM = Xsize;
  int colsM = Ysize;

  // pitchM: leading dimension of M, which is ideally equal to rowsM,
  //          but actually larger than that.
  int pitchM = (rowsM / 4 + 1) * 4;

  memCUDA(M, pitchM*colsM);

  // fprintf(stderr, "rowsM, rowsM*sizeof(float), colsM : %d %d %d\n",
  // 	  rowsM, rowsM * sizeof(float), colsM)


  memCUDA(D, 3*rowsM); // temporary vector


  // a vector with all elements of 1.0f
  float* h_one = new float [max(Xsize,Ysize)];
  for(int t = 0; t < max(Xsize,Ysize); t++) h_one[t] = 1.0f;
  memHostToCUDA(one, max(Xsize,Ysize)); // vector with all elements of 1


  memCUDA(sumOfMRow, rowsM);
  memCUDA(sumOfMCol, colsM);

  float* h_m_outliers_row = new float [rowsM]; 
  float* h_m_outliers_col = new float [colsM];
  for(int i = 0; i < rowsM; i++) h_m_outliers_row[i] = moutlier;
  for(int i = 0; i < colsM; i++) h_m_outliers_col[i] = moutlier;
  memHostToCUDA(m_outliers_row, rowsM);
  memHostToCUDA(m_outliers_col, colsM);


  

  //
  // threads
  //


  // for 2D block
  dim3 dimBlockForM(BLOCK_SIZE, BLOCK_SIZE); // a block is (BLOCK_SIZE*BLOCK_SIZE) threads
  dim3 dimGridForM( (pitchM + dimBlockForM.x - 1) / dimBlockForM.x,
		    (colsM  + dimBlockForM.y - 1) / dimBlockForM.y);

  // for 1D block
  int threadsPerBlockForYsize = 512; // a block is 512 threads
  int blocksPerGridForYsize
    = (Ysize + threadsPerBlockForYsize - 1 ) / threadsPerBlockForYsize;
  int threadsPerBlockForXsize = 512; // a block is 512 threads
  int blocksPerGridForXsize
    = (Xsize + threadsPerBlockForXsize - 1 ) / threadsPerBlockForYsize;







  //
  // timer
  //


#define START_TIMER(timer) \
  if(!param.notimer){ \
      CUDA_SAFE_CALL( hipDeviceSynchronize() );\
      CUT_SAFE_CALL(cutStartTimer(timer)); \
  }
#define STOP_TIMER(timer) \
  if(!param.notimer){ \
      CUDA_SAFE_CALL( hipDeviceSynchronize() );\
      CUT_SAFE_CALL(cutStopTimer(timer)); \
  }


  // timers
  unsigned int timerTotal, 
    timerUpdateM, timerShinkhorn, timerSumM,
    timerGetWeightedXY, timerGetXcYc, timerCenteringXY, timerFindS, timerAfterSVD, timerRT,
    timerShinkhorn1, timerShinkhorn2, timerShinkhorn3;

  if(!param.notimer){
    CUT_SAFE_CALL(cutCreateTimer(&timerUpdateM));
    CUT_SAFE_CALL(cutCreateTimer(&timerShinkhorn));
    CUT_SAFE_CALL(cutCreateTimer(&timerShinkhorn1));
    CUT_SAFE_CALL(cutCreateTimer(&timerShinkhorn2));
    CUT_SAFE_CALL(cutCreateTimer(&timerShinkhorn3));
    CUT_SAFE_CALL(cutCreateTimer(&timerSumM));
    CUT_SAFE_CALL(cutCreateTimer(&timerGetWeightedXY));
    CUT_SAFE_CALL(cutCreateTimer(&timerGetXcYc));
    CUT_SAFE_CALL(cutCreateTimer(&timerCenteringXY));
    CUT_SAFE_CALL(cutCreateTimer(&timerFindS));
    CUT_SAFE_CALL(cutCreateTimer(&timerAfterSVD));
    CUT_SAFE_CALL(cutCreateTimer(&timerRT));
  }


  CUT_SAFE_CALL(cutCreateTimer(&timerTotal));
  CUDA_SAFE_CALL( hipDeviceSynchronize() );
  CUT_SAFE_CALL(cutStartTimer(timerTotal));




  //
  // initializing cublas
  //
  cublasInit();



  //
  // softassign main loop
  //

  for(int Titer = 1; Titer <= JMAX; Titer++){

    fprintf(stderr, "%d iter. temp. %f  ", Titer, T_cur);
    fprintf(stderr, "time %.10f [s]\n", cutGetTimerValue(timerTotal) / 1000.0f);

#ifndef NOVIEWER
    if(!param.noviewer){
      UpdatePointCloud2(Ysize, param.points2, h_Y, h_R, h_t);
      if (!EngineIteration()) // PointCloudViewer
	break;
    }
#endif


    // inner loop with the same temperature

    for(int iter0 = 0; iter0 < I0; iter0++){

      //
      // UpdateM
      //

      START_TIMER(timerUpdateM);

      updateM
	<<< dimGridForM, dimBlockForM >>>
	(rowsM, colsM, pitchM,
	 d_Xx, d_Xy, d_Xz,
	 d_Yx, d_Yy, d_Yz,
	 d_R, d_t, d_M, T_cur, alpha);

      STOP_TIMER(timerUpdateM);




      //
      // Normalization of M by Shinkhorn
      //


      START_TIMER(timerShinkhorn);

      // shinkhorn loop until M converges
      for (int Sinkh_iter = 0; Sinkh_iter < I1; Sinkh_iter++){


	//
	// row normalization
	//

	START_TIMER(timerShinkhorn1);

	// hipblasSgemv (char trans, int m, int n, float alpha, const float *A, int lda,
	//              const float *x, int incx, float beta, float *y, int incy)
	//    y = alpha * op(A) * x + beta * y,
	
	// M * one vector = vector with elements of row-wise sum
	//     d_M      *    d_one    =>  d_sumOfMRow
	//(rowsM*colsM) *  (colsM*1)  =  (rowsM*1)
	hipblasSgemv('n',          // char trans
		    rowsM, colsM, // int m (rows of A), n (cols of A) ; not op(A)
		    1.0f,         // float alpha
		    d_M, pitchM,  // const float *A, int lda
		    d_one, 1,     // const float *x, int incx
		    0.0f,         // float beta
		    d_sumOfMRow, 1);   // float *y, int incy

	STOP_TIMER(timerShinkhorn1);
	START_TIMER(timerShinkhorn2);

	// void hipblasSaxpy (int n, float alpha, const float *x, int incx, float *y, int incy)
	// alpha * x + y => y
	// m_outliers_row + d_sumOfMRow => d_sumOfMRow
	hipblasSaxpy(rowsM, 1.0f, d_m_outliers_row, 1, d_sumOfMRow, 1);


	STOP_TIMER(timerShinkhorn2);
	START_TIMER(timerShinkhorn3);

	normalizeMbySinkhorn_row
	  <<< dimGridForM, dimBlockForM >>>
	  (rowsM, colsM, pitchM,
	   d_M, d_sumOfMRow, d_m_outliers_row);


	STOP_TIMER(timerShinkhorn3);



	//
	// column normalization
	//

	// hipblasSgemv (char trans, int m, int n, float alpha, const float *A, int lda,
	//              const float *x, int incx, float beta, float *y, int incy)
	//    y = alpha * op(A) * x + beta * y,
	
	// M * one vector = vector with elements of column-wise sum
	//     d_M^T    *    d_one    =>  d_sumOfMCol
	//(coslM*rowsM) *  (rowsM*1)  =  (colsM*1)
	hipblasSgemv('t',          // char trans
		    rowsM, colsM, // int m (rows of A), n (cols of A) ; not op(A)
		    1.0f,         // float alpha
		    d_M, pitchM,  // const float *A, int lda
		    d_one, 1,     // const float *x, int incx
		    0.0f,         // float beta
		    d_sumOfMCol, 1);   // float *y, int incy

	// void hipblasSaxpy (int n, float alpha, const float *x, int incx, float *y, int incy)
	// alpha * x + y => y
	// m_outliers_col + d_sumOfMCol => d_sumOfMCol
	hipblasSaxpy(colsM, 1.0f, d_m_outliers_col, 1, d_sumOfMCol, 1);

	

	normalizeMbySinkhorn_col
	  <<< dimGridForM, dimBlockForM >>> 
	  (rowsM, colsM, pitchM,
	   d_M, d_sumOfMCol, d_m_outliers_col);

      }

      STOP_TIMER(timerShinkhorn);




      //
      // update R,T
      //


      ///////////////////////////////////////////////////////////////////////////////////// 

      // compute sum of all elements in M


      START_TIMER(timerSumM);

      // hipblasSgemv (char trans, int m, int n, float alpha, const float *A, int lda,
      //              const float *x, int incx, float beta, float *y, int incy)
      //    y = alpha * op(A) * x + beta * y,

      // M * one vector = vector with elements of row-wise sum
      //     d_M      *    d_one    =>  d_sumOfMRow
      //(rowsM*colsM) *  (colsM*1)  =  (rowsM*1)

      hipblasSgemv('n',          // char trans
		  rowsM, colsM, // int m (rows of A), n (cols of A) ; not op(A)
		  1.0f,         // float alpha
		  d_M, pitchM,  // const float *A, int lda
		  d_one, 1,     // const float *x, int incx
		  0.0f,         // float beta
		  d_sumOfMRow, 1);   // float *y, int incy
      


      //sum of M
      // float hipblasSasum (int n, const float *x, int incx) 
      // computes the sum of the absolute values of the elements
      float sumM = hipblasSasum (rowsM, d_sumOfMRow, 1); 
      // sum of all elements in M, assuming that all are positive.

      STOP_TIMER(timerSumM);


      ///////////////////////////////////////////////////////////////////////////////////// 


      // compute weighted X and Y
	
      START_TIMER(timerGetWeightedXY);

      // X .* sumOfRow => X_result
      elementwiseMultiplicationCopy
	<<< blocksPerGridForXsize, threadsPerBlockForXsize>>>
	(rowsM,
	 d_Xx, d_Xy, d_Xz,
	 d_sumOfMRow,
	 d_Xx_result, d_Xy_result, d_Xz_result);

      // Y .* sumOfCol => Y_result
      elementwiseMultiplicationCopy
	<<< blocksPerGridForYsize, threadsPerBlockForYsize>>>
	(colsM,
	 d_Yx, d_Yy, d_Yz,
	 d_sumOfMCol,
	 d_Yx_result, d_Yy_result, d_Yz_result);
      
      STOP_TIMER(timerGetWeightedXY);


      ///////////////////////////////////////////////////////////////////////////////////// 


      // find weighted center of X' and Y

      START_TIMER(timerGetXcYc);

      // hipblasSasum can not be used for summing up a vector
      //  because it is ABS sum, not just sum.

      // hipblasSgemv (char trans, int m, int n, float alpha, const float *A, int lda,
      //              const float *x, int incx, float beta, float *y, int incy)
      //    y = alpha * op(A) * x + beta * y,

      // d_X_result^T *    d_one     =>   h_Xc
      //  (3 * rowsM)   (rowsM * 1)  =  (3 * 1)
      hipblasSgemv('t',          // char trans
		  rowsM, 3,     // int m (rows of A), n (cols of A) ; not op(A)
		  1.0f,         // float alpha
		  d_X_result, rowsM,  // const float *A, int lda
		  d_one, 1,     // const float *x, int incx
		  0.0f,         // float beta
		  d_Xc, 1);     // float *y, int incy

      // d_Y_result^T *    d_one     =>   h_Yc
      //  (3 * colsM)   (colM * 1)  =  (3 * 1)
      hipblasSgemv('t',          // char trans
		  colsM, 3,     // int m (rows of A), n (cols of A) ; not op(A)
		  1.0f,         // float alpha
		  d_Y_result, colsM,  // const float *A, int lda
		  d_one, 1,     // const float *x, int incx
		  0.0f,         // float beta
		  d_Yc, 1);     // float *y, int incy


      // void hipblasSscal (int n, float alpha, float *x, int incx)
      // it replaces x[ix + i * incx] with alpha * x[ix + i * incx]
      hipblasSscal (3, 1/sumM, d_Xc, 1);
      hipblasSscal (3, 1/sumM, d_Yc, 1);


      CUDA_SAFE_CALL(hipMemcpy(h_Xc, d_Xc, sizeof(float)*3, hipMemcpyDeviceToHost));
      CUDA_SAFE_CALL(hipMemcpy(h_Yc, d_Yc, sizeof(float)*3, hipMemcpyDeviceToHost));


      STOP_TIMER(timerGetXcYc);


      ///////////////////////////////////////////////////////////////////////////////////// 

      // centering X and Y

      START_TIMER(timerCenteringXY);

      centeringXorY
	<<< blocksPerGridForXsize, threadsPerBlockForXsize>>>
	(rowsM,
	 d_Xc, sumM,
	 d_Xx_result, d_Xy_result, d_Xz_result);

      centeringXorY
	<<< blocksPerGridForYsize, threadsPerBlockForYsize>>>
	(colsM,
	 d_Yc, sumM,
	 d_Yx_result, d_Yy_result, d_Yz_result);
 

      STOP_TIMER(timerCenteringXY);


      ///////////////////////////////////////////////////////////////////////////////////// 

      // compute S

      START_TIMER(timerFindS)

      // S = d_X_result^T * d_M * d_Y_result

      // hipblasSgemm (char transa, char transb, int m, int n, int k, float alpha, 
      //              const float *A, int lda, const float *B, int ldb, float beta, 
      //              float *C, int ldc)
      //   C = alpha * op(A) * op(B) + beta * C,
      //
      // m      number of rows of matrix op(A) and rows of matrix C
      // n      number of columns of matrix op(B) and number of columns of C
      // k      number of columns of matrix op(A) and number of rows of op(B) 

      //     d_M      * d_Y_result =>    d_D
      //(rowsM*colsM) *  (colsM*3)  =  (rowsM*3)
      //   m  * k           k * n        m * n   
      hipblasSgemm('n', 'n', rowsM, 3, colsM, 
		  1.0f, d_M, pitchM,
		  d_Y_result, colsM,
		  0.0f, d_D, rowsM);

      //  d_X_result^T *     d_D     =>  d_S
      //    (3*rowsM)  *  (rowsM*3)  =  (3*3)
      //   m  * k           k * n        m * n
      hipblasSgemm('t', 'n', 3, 3, rowsM,
		  1.0f, d_X_result, rowsM,
		  d_D, rowsM,
		  0.0f, d_S, 3);

      CUDA_SAFE_CALL(hipMemcpy(h_S, d_S, sizeof(float)*9, hipMemcpyDeviceToHost));


      STOP_TIMER(timerFindS);



      ///////////////////////////////////////////////////////////////////////////////////// 

      // find RT from S

      START_TIMER(timerAfterSVD);

      findRTfromS(h_Xc, h_Yc, h_S, h_R, h_t, false);

      STOP_TIMER(timerAfterSVD);

      ///////////////////////////////////////////////////////////////////////////////////// 

      // copy R,t to device

      START_TIMER(timerRT);

      CUDA_SAFE_CALL(hipMemcpy(d_R, h_R, sizeof(float)*3*3, hipMemcpyHostToDevice));
      CUDA_SAFE_CALL(hipMemcpy(d_t, h_t, sizeof(float)*3,   hipMemcpyHostToDevice));

      STOP_TIMER(timerRT);

      ///////////////////////////////////////////////////////////////////////////////////// 

#ifndef NOVIEWER
      if(!param.noviewer){
	UpdatePointCloud2(Ysize, param.points2, h_Y, h_R, h_t);
	if (!EngineIteration()) // PointCloudViewer
	  break;
      }
#endif

    }

    T_cur = T_cur*TFACTR;

  }


  CUDA_SAFE_CALL( hipDeviceSynchronize() );
  CUT_SAFE_CALL(cutStopTimer(timerTotal));

  fprintf(stderr, "comping time: %.10f [s]\n", cutGetTimerValue(timerTotal) / 1000.0f);

  if(!param.notimer){

    fprintf(stderr, "Average %.10f [s] for %s\n", cutGetAverageTimerValue(timerUpdateM)  / 1000.0f, "updateM");
    fprintf(stderr, "Average %.10f [s] for %s\n", cutGetAverageTimerValue(timerShinkhorn)/ 1000.0f, "shinkhorn");
    fprintf(stderr, "Average %.10f [s] for %s\n", cutGetAverageTimerValue(timerShinkhorn1)/ 1000.0f, "shinkhorn1");
    fprintf(stderr, "Average %.10f [s] for %s\n", cutGetAverageTimerValue(timerShinkhorn2)/ 1000.0f, "shinkhorn2");
    fprintf(stderr, "Average %.10f [s] for %s\n", cutGetAverageTimerValue(timerShinkhorn3)/ 1000.0f, "shinkhorn3");
    fprintf(stderr, "Average %.10f [s] for %s\n", cutGetAverageTimerValue(timerSumM)  / 1000.0f, "SumM");
    fprintf(stderr, "Average %.10f [s] for %s\n", cutGetAverageTimerValue(timerGetWeightedXY)   / 1000.0f, "getMXY");
    fprintf(stderr, "Average %.10f [s] for %s\n", cutGetAverageTimerValue(timerGetXcYc)  / 1000.0f, "getXcYc");
    fprintf(stderr, "Average %.10f [s] for %s\n", cutGetAverageTimerValue(timerCenteringXY) / 1000.0f, "getNewXY");
    fprintf(stderr, "Average %.10f [s] for %s\n", cutGetAverageTimerValue(timerFindS)    / 1000.0f, "findS");
    fprintf(stderr, "Average %.10f [s] for %s\n", cutGetAverageTimerValue(timerAfterSVD) / 1000.0f, "afterSVD");
    fprintf(stderr, "Average %.10f [s] for %s\n", cutGetAverageTimerValue(timerRT) / 1000.0f, "RT");

    CUT_SAFE_CALL(cutDeleteTimer(timerTotal));
    CUT_SAFE_CALL(cutDeleteTimer(timerUpdateM));
    CUT_SAFE_CALL(cutDeleteTimer(timerShinkhorn));
    CUT_SAFE_CALL(cutDeleteTimer(timerShinkhorn1));
    CUT_SAFE_CALL(cutDeleteTimer(timerShinkhorn2));
    CUT_SAFE_CALL(cutDeleteTimer(timerShinkhorn3));
    CUT_SAFE_CALL(cutDeleteTimer(timerSumM));
    CUT_SAFE_CALL(cutDeleteTimer(timerGetWeightedXY));
    CUT_SAFE_CALL(cutDeleteTimer(timerGetXcYc));
    CUT_SAFE_CALL(cutDeleteTimer(timerCenteringXY));
    CUT_SAFE_CALL(cutDeleteTimer(timerFindS));
    CUT_SAFE_CALL(cutDeleteTimer(timerAfterSVD));

  }

  cublasShutdown();




  CUDA_SAFE_CALL(hipFree(d_Xx));
  CUDA_SAFE_CALL(hipFree(d_Yx));
  CUDA_SAFE_CALL(hipFree(d_R));
  CUDA_SAFE_CALL(hipFree(d_t));
  CUDA_SAFE_CALL(hipFree(d_M));
  CUDA_SAFE_CALL(hipFree(d_D));
  CUDA_SAFE_CALL(hipFree(d_S));
  CUDA_SAFE_CALL(hipFree(d_one));
  CUDA_SAFE_CALL(hipFree(d_sumOfMRow));
  CUDA_SAFE_CALL(hipFree(d_sumOfMCol));
  CUDA_SAFE_CALL(hipFree(d_X_result));
  CUDA_SAFE_CALL(hipFree(d_Y_result));
  CUDA_SAFE_CALL(hipFree(d_m_outliers_row));
  CUDA_SAFE_CALL(hipFree(d_m_outliers_col));   

  delete [] h_m_outliers_row;
  delete [] h_m_outliers_col; 

  delete [] h_one;

  CUDA_SAFE_CALL( hipDeviceReset() );
}
