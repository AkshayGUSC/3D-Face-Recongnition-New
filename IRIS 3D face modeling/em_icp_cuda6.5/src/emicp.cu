#include "hip/hip_runtime.h"
/*
  Copyright (c) 2010 Toru Tamaki

  Permission is hereby granted, free of charge, to any person
  obtaining a copy of this software and associated documentation
  files (the "Software"), to deal in the Software without
  restriction, including without limitation the rights to use,
  copy, modify, merge, publish, distribute, sublicense, and/or sell
  copies of the Software, and to permit persons to whom the
  Software is furnished to do so, subject to the following
  conditions:

  The above copyright notice and this permission notice shall be
  included in all copies or substantial portions of the Software.

  THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,
  EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES
  OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND
  NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT
  HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY,
  WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
  FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR
  OTHER DEALINGS IN THE SOFTWARE.
*/


#include "cudaMem.h"

#include <iostream>
#include <algorithm>
#include <cstdio>

//cuda
#include <hip/hip_runtime_api.h>
#include <helper_cuda_drvapi.h>
#include <helper_cuda_gl.h>
#include <helper_timer.h>
#include <helper_string.h>
#include <hip/hip_vector_types.h>
#include <helper_image.h>
#include <helper_functions.h>
#include <>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include "hipblas.h"

// uncomment if you do not use the viewer.
//#define NOVIEWER

#include "3dregistration.h"
#include "engine.h"

using namespace std;





	

/***************************/
__global__ static void
d_updateA(int rowsA, int colsA, int pitchA,
	const float* d_Xx, const float* d_Xy, const float* d_Xz, 
	const float* d_Yx, const float* d_Yy, const float* d_Yz,
	const float* d_R, const float* d_t,
	float* d_A,
	float sigma_p2){


  int r =  blockIdx.x * blockDim.x + threadIdx.x;
  int c =  blockIdx.y * blockDim.y + threadIdx.y;

  // Shared memory
  __shared__ float XxShare[BLOCK_SIZE];
  __shared__ float XyShare[BLOCK_SIZE];
  __shared__ float XzShare[BLOCK_SIZE];
  __shared__ float YxShare[BLOCK_SIZE];
  __shared__ float YyShare[BLOCK_SIZE];
  __shared__ float YzShare[BLOCK_SIZE];
  __shared__ float RShare[9]; // BLOCK_SIZE >= 9 is assumed
  __shared__ float tShare[3]; // BLOCK_SIZE >= 3 is assumed

  if(threadIdx.y == 0)
    if(// 0 <= threadIdx.x &&  // threadIdx.x is unsigned int, so always positive
       threadIdx.x < 9){
      RShare[threadIdx.x] = d_R[threadIdx.x];
      if(threadIdx.x < 3)
	tShare[threadIdx.x] = d_t[threadIdx.x];
    }

  if(r < rowsA && c < colsA){ // check for only inside the matrix A

    if(threadIdx.x == 0){
      XxShare[threadIdx.y] = d_Xx[c];
      XyShare[threadIdx.y] = d_Xy[c];
      XzShare[threadIdx.y] = d_Xz[c];
    }
    if(threadIdx.y == 0){
      YxShare[threadIdx.x] = d_Yx[r];
      YyShare[threadIdx.x] = d_Yy[r];
      YzShare[threadIdx.x] = d_Yz[r];
    }

    __syncthreads();

#define Xx XxShare[threadIdx.y]
#define Xy XyShare[threadIdx.y]
#define Xz XzShare[threadIdx.y]
#define Yx YxShare[threadIdx.x]
#define Yy YyShare[threadIdx.x]
#define Yz YzShare[threadIdx.x]
#define R(i) RShare[i]
#define t(i) tShare[i]

// #define Euclid(a,b,c) ((a)*(a)+(b)*(b)+(c)*(c))
//     float tmp =
//       Euclid(Xx - (R(0)*Yx + R(1)*Yy + R(2)*Yz + t(0)),
//              Xy - (R(3)*Yx + R(4)*Yy + R(5)*Yz + t(1)),
//              Xz - (R(6)*Yx + R(7)*Yy + R(8)*Yz + t(2)) );
    
//     tmp = expf(-tmp/sigma_p^2)


     float tmpX = Xx - (R(0)*Yx + R(1)*Yy + R(2)*Yz + t(0));
     float tmpY = Xy - (R(3)*Yx + R(4)*Yy + R(5)*Yz + t(1));
     float tmpZ = Xz - (R(6)*Yx + R(7)*Yy + R(8)*Yz + t(2));

    __syncthreads();

     tmpX *= tmpX;
     tmpY *= tmpY;
     tmpZ *= tmpZ;

     tmpX += tmpY;
     tmpX += tmpZ;

     tmpX /= sigma_p2;
     tmpX = expf(-tmpX);


    //float *A = (float*)((char*)d_A + c * pitchMinBytes) + r;

    d_A[c * pitchA + r] = tmpX;
  }

}
/***************************/

/***************************/
__global__ static void
d_normalizeRowsOfA(int rowsA, int colsA, int pitchA,
		 float *d_A,
		 const float *d_C
		 ){
  
  int r =  blockIdx.x * blockDim.x + threadIdx.x;
  int c =  blockIdx.y * blockDim.y + threadIdx.y;

  // Shared memory
  __shared__ float d_CShare[BLOCK_SIZE];


  if(r < rowsA && c < colsA){ // check for only inside the matrix A

    if(threadIdx.y == 0)
      d_CShare[threadIdx.x] = d_C[r];

    __syncthreads();

    if(d_CShare[threadIdx.x] > 10e-7f)
      // each element in A is normalized C, then squre-rooted
      d_A[c * pitchA + r] = sqrtf( d_A[c * pitchA + r] / d_CShare[threadIdx.x] );
    else
      d_A[c * pitchA + r] = 1.0f/colsA; // ad_hoc code to avoid 0 division

    __syncthreads();

  }

}
/***************************/

/***************************/
__global__ static void
d_elementwiseDivision(int Xsize,
		    float* d_Xx, float* d_Xy, float* d_Xz,
		    const float* d_lambda){

  int x =  blockIdx.x * blockDim.x + threadIdx.x;

  if(x < Xsize){ // check for only inside X
    float l_lambda = d_lambda[x];
    d_Xx[x] /= l_lambda;
    d_Xy[x] /= l_lambda;
    d_Xz[x] /= l_lambda;
  }
}
/***************************/

/***************************/
__global__ static void
d_elementwiseMultiplication(int Xsize,
			  float* d_Xx, float* d_Xy, float* d_Xz,
			  const float* d_lambda){

  int x =  blockIdx.x * blockDim.x + threadIdx.x;

  if(x < Xsize){ // check for only inside X
    float l_lambda = d_lambda[x];
    d_Xx[x] *= l_lambda;
    d_Xy[x] *= l_lambda;
    d_Xz[x] *= l_lambda;
  }
}
/***************************/

/***************************/
__global__ static void
d_centeringXandY(int rowsA,
	       const float* d_Xc, const float* d_Yc,
	       const float* d_Xx, const float* d_Xy, const float* d_Xz,
	       const float* d_Yx, const float* d_Yy, const float* d_Yz,
	       float* d_XxCenterd, float* d_XyCenterd, float* d_XzCenterd,
	       float* d_YxCenterd, float* d_YyCenterd, float* d_YzCenterd
	       ){

  // do for both X and Y at the same time
  
  int r =  blockIdx.x * blockDim.x + threadIdx.x;

  // Shared memory
  __shared__ float Xc[3];
  __shared__ float Yc[3];

  if(threadIdx.x < 6) // assume blocksize >= 6
    if(threadIdx.x < 3) 
      Xc[threadIdx.x] = d_Xc[threadIdx.x];
    else
      Yc[threadIdx.x - 3] = d_Yc[threadIdx.x - 3];


  if(r < rowsA){ // check for only inside the vectors

    __syncthreads();

    d_XxCenterd[r] = d_Xx[r] - Xc[0];
    d_XyCenterd[r] = d_Xy[r] - Xc[1];
    d_XzCenterd[r] = d_Xz[r] - Xc[2];

    d_YxCenterd[r] = d_Yx[r] - Yc[0];
    d_YyCenterd[r] = d_Yy[r] - Yc[1];
    d_YzCenterd[r] = d_Yz[r] - Yc[2];

    __syncthreads();

  }
}
/***************************/









/************************/
/*		EM-ICP function	*/
void emicp(	int Xsize, int Ysize,
			float *h_X, float *d_X, float *d_Xx, float *d_Xy, float *d_Xz,
			float* h_Y, float *d_Y, float *d_Yx, float *d_Yy, float *d_Yz,
			float* h_R, float *d_R, float* h_t, float *d_t,
			float* h_S, float *d_S,
			float *h_Xc, float *d_Xc, float *h_Yc, float *d_Yc,
			float *h_one, float *d_one,
			float *d_A,
			float *d_Xprime, float *d_XprimeX, float *d_XprimeY, float *d_XprimeZ,
			float *d_XprimeCenterd, float *d_XprimeCenterdX, float *d_XprimeCenterdY, float *d_XprimeCenterdZ,
			float *d_YCenterd, float *d_YCenterdX, float *d_YCenterdY, float *d_YCenterdZ,
			float *d_C, float *d_lambda,
			int	maxXY, int rowsA, int colsA, int pitchA,
			registrationParameters param,
			bool *error,
			bool *allocateMemory
	   ){

	
	//
	// initialize paramters
	//
	float sigma_p2 = param.sigma_p2;
	float sigma_inf = param.sigma_inf;
	float sigma_factor = param.sigma_factor;
	float d_02 = param.d_02;


	// pitchA:	leading dimension of A, which is ideally equal to rowsA,
	//          but actually larger than that.

	//
	// memory allocation
	//
	

	//
	// initializing CUDA
	//
	// CUT_DEVICE_INIT(param.argc, param.argv);	
	
	
	// R, t
	//copyHostToCUDA(R, 9);
	//copyHostToCUDA(t, 3);
	
	

	//for (int j=0; j<9; j++)
	//	printf("%f %f\n", h_R[j], d_R[j]);


	// NOTE on matrix A
	// number of rows:     Ysize, or rowsA
	// number of columns : Xsize, or colsA
	// 
	//                    [0th in X] [1st]  ... [(Xsize-1)] 
	// [0th point in Y] [ A(0,0)     A(0,1) ... A(0,Xsize-1)      ] 
	// [1st           ] [ A(1,0)     A(1,1) ...                   ]
	// ...              [ ...                                     ]
	// [(Ysize-1)     ] [ A(Ysize-1, 0)     ... A(Ysize-1,Xsize-1)]
	//
	// 
	// CAUTION on matrix A
	// A is allcoated as a column-maijor format for the use of cublas.
	// This means that you must acces an element at row r and column c as:
	// A(r,c) = A[c * pitchA + r]



	//
	// threads
	//

	// for 2D block
	dim3 dimBlockForA(BLOCK_SIZE, BLOCK_SIZE); // a block is (BLOCK_SIZE*BLOCK_SIZE) threads
	dim3 dimGridForA( (pitchA + dimBlockForA.x - 1) / dimBlockForA.x,
			 (colsA  + dimBlockForA.y - 1) / dimBlockForA.y);

	// for 1D block
	int threadsPerBlockForYsize = ICP_CUDA_BLOCK; // a block is 512 threads
	int blocksPerGridForYsize
	 = (Ysize + threadsPerBlockForYsize - 1 ) / threadsPerBlockForYsize;


	//
	// timer
	//

	// timers
	//unsigned int timerUpdateA, timerAfterSVD, timerRT;


	//if(!param.notimer){
	// CUT_SAFE_CALL(cutCreateTimer(&timerUpdateA));
	// CUT_SAFE_CALL(cutCreateTimer(&timerAfterSVD));
	// CUT_SAFE_CALL(cutCreateTimer(&timerRT));
	//}


	//CUT_SAFE_CALL(	cutCreateTimer(&timerTotal));
	//CUDA_SAFE_CALL( hipDeviceSynchronize() );
	//CUT_SAFE_CALL(	cutStartTimer(timerTotal));




	//////////////////////////////////////////////////////////////////////////////////////////
	//																						//
	//												EM-ICP main loop						//
	//																						//
	//////////////////////////////////////////////////////////////////////////////////////////
	float pre_Xc[3], pre_Yc[3];

	while(sigma_p2 > sigma_inf){
		// Remember Xc, Yc
		for (int i=0; i<3; i++) {
			pre_Xc[i] = h_Xc[i];
			pre_Yc[i] = h_Yc[i];
		}

		copyHostToCUDA(R,9);
		copyHostToCUDA(t,3);

		//fprintf(stderr, "%d iter. sigma_p2 %f  ", Titer++, sigma_p2);
		//fprintf(stderr, "time %.10f [s]\n", cutGetTimerValue(timerTotal) / 1000.0f);

#ifndef NOVIEWER
	if(!param.noviewer)
		if (!EngineIteration()) // PointCloudViewer
			break;
#endif

		//
		// UpdateA
		//

		//START_TIMER(timerUpdateA);

		d_updateA <<< dimGridForA, dimBlockForA >>>
			(rowsA, colsA, pitchA,
			 d_Xx, d_Xy, d_Xz, 
			 d_Yx, d_Yy, d_Yz,
			 d_R, d_t, 
			 d_A, sigma_p2);

		//STOP_TIMER(timerUpdateA);


		//
		// Normalization of A
		//

		// hipblasSgemv (char trans, int m, int n, float alpha, const float *A, int lda,
		//              const float *x, int incx, float beta, float *y, int incy)
		//    y = alpha * op(A) * x + beta * y,
      
		// A * one vector = vector with elements of row-wise sum
		//     d_A      *    d_one    =>  d_C
		//(rowsA*colsA) *  (colsA*1)  =  (rowsA*1)
		hipblasSgemv(	'n',          // char trans
							rowsA, colsA, // int m (rows of A), n (cols of A) ; not op(A)
							1.0f,         // float alpha
							d_A, pitchA,  // const float *A, int lda
							d_one, 1,     // const float *x, int incx
							0.0f,         // float beta
							d_C, 1);      // float *y, int incy


		// void hipblasSaxpy (int n, float alpha, const float *x, int incx, float *y, int incy)
		// alpha * x + y => y
		// exp(-d_0^2/sigma_p2) * d_one + d_C => d_C
		float xp = expf(-d_02/sigma_p2);
		hipblasSaxpy(rowsA, xp, d_one, 1, d_C, 1);
      
		d_normalizeRowsOfA	<<< dimGridForA, dimBlockForA >>>
			(rowsA, colsA, pitchA, d_A, d_C);


		//
		// update R,T
		//

		///////////////////////////////////////////////////////////////////////////////////// 
		// compute lambda
      
		// A * one vector = vector with elements of row-wise sum
		//     d_A      *    d_one    =>  d_lambda
		//(rowsA*colsA) *  (colsA*1)  =  (rowsA*1)
		
		hipblasSgemv(	'n',          // char trans
							rowsA, colsA, // int m (rows of A), n (cols of A) ; not op(A)
							1.0f,         // float alpha
							d_A, pitchA,  // const float *A, int lda
							d_one, 1,     // const float *x, int incx
							0.0f,         // float beta
							d_lambda, 1); // float *y, int incy
		


		// float hipblasSasum (int n, const float *x, int incx) 
		float sumLambda = hipblasSasum (rowsA, d_lambda, 1);


		///////////////////////////////////////////////////////////////////////////////////// 
		// compute X'

		// hipblasSgemm (char transa, char transb, int m, int n, int k, float alpha, 
		//              const float *A, int lda, const float *B, int ldb, float beta, 
		//              float *C, int ldc)
		//   C = alpha * op(A) * op(B) + beta * C,
		//
		// m      number of rows of matrix op(A) and rows of matrix C
		// n      number of columns of matrix op(B) and number of columns of C
		// k      number of columns of matrix op(A) and number of rows of op(B) 

		// A * X => X'
		//     d_A      *    d_X    =>  d_Xprime
		//(rowsA*colsA) *  (colsA*3)  =  (rowsA*3)
		//   m  * k           k * n        m * n   
		hipblasSgemm(	'n', 'n', rowsA, 3, colsA,
							1.0f, d_A, pitchA,
							d_X, colsA,
							0.0f, d_Xprime, rowsA);


		// X' ./ lambda => X'
		d_elementwiseDivision 	<<< blocksPerGridForYsize, threadsPerBlockForYsize>>>
			(rowsA, d_XprimeX, d_XprimeY, d_XprimeZ, d_lambda);


		///////////////////////////////////////////////////////////////////////////////////// 
		//
		// centering X' and Y
		//

		///////////////////////////////////////////////////////////////////////////////////// 
		// find weighted center of X' and Y

		// d_Xprime^T *    d_lambda     =>   h_Xc
		//  (3 * rowsA)   (rowsA * 1)  =  (3 * 1)
		hipblasSgemv('t',					// char trans
						rowsA, 3,			// int m (rows of A), n (cols of A) ; not op(A)
						1.0f,					// float alpha
						d_Xprime, rowsA,  // const float *A, int lda
						d_lambda, 1,		// const float *x, int incx
						0.0f,					// float beta
						d_Xc, 1);			// float *y, int incy

		// d_Y^T *    d_lambda     =>   h_Yc
		//  (3 * rowsA)   (rowsA * 1)  =  (3 * 1)
		hipblasSgemv('t',				// char trans
						rowsA, 3,		// int m (rows of A), n (cols of A) ; not op(A)
						1.0f,				// float alpha
						d_Y, rowsA,		// const float *A, int lda
						d_lambda, 1,	// const float *x, int incx
						0.0f,				// float beta
						d_Yc, 1);		// float *y, int incy

		// void hipblasSscal (int n, float alpha, float *x, int incx)
		// it replaces x[ix + i * incx] with alpha * x[ix + i * incx]
		float invSumLambda = 1/sumLambda;
		hipblasSscal (3, invSumLambda, d_Xc, 1);
		hipblasSscal (3, invSumLambda, d_Yc, 1);

		


		///////////////////////////////////////////////////////////////////////////////////// 
		// centering X and Y

		// d_Xprime .- d_Xc => d_XprimeCenterd
		// d_Y      .- d_Yc => d_YCenterd
		d_centeringXandY	<<< blocksPerGridForYsize, threadsPerBlockForYsize>>>
			(rowsA, 
			 d_Xc, d_Yc,
			 d_XprimeX, d_XprimeY, d_XprimeZ,
			 d_Yx, d_Yy, d_Yz,
			 d_XprimeCenterdX, d_XprimeCenterdY, d_XprimeCenterdZ,
			 d_YCenterdX, d_YCenterdY, d_YCenterdZ);

		// XprimeCented .* d_lambda => XprimeCented
		d_elementwiseMultiplication	<<< blocksPerGridForYsize, threadsPerBlockForYsize >>>
			(rowsA, d_XprimeCenterdX, d_XprimeCenterdY, d_XprimeCenterdZ, d_lambda);

		///////////////////////////////////////////////////////////////////////////////////// 
		// compute S

		//  d_XprimeCented^T *   d_YCenterd     =>  d_S
		//    (3*rowsA)  *  (rowsA*3)  =  (3*3)
		//   m  * k           k * n        m * n
		hipblasSgemm('t', 'n', 3, 3, rowsA,
						1.0f, d_XprimeCenterd, rowsA,
						d_YCenterd, rowsA,
						0.0f, d_S, 3);

		

		///////////////////////////////////////////////////////////////////////////////////// 
		// find RT from S

		//START_TIMER(timerAfterSVD);
		copyCUDAToHost(S,9);
		copyCUDAToHost(Xc,3);
		copyCUDAToHost(Yc,3);

		// Remember the latest value in case of failure
		#define h_Xcx h_Xc[0]
		#define h_Xcy h_Xc[1]
		#define h_Xcz h_Xc[2]
		#define h_Ycx h_Yc[0]
		#define h_Ycy h_Yc[1]
		#define h_Ycz h_Yc[2]		
		if (h_Xcx != h_Xcx || h_Xcy != h_Xcy || h_Xcz != h_Xcz) {
			for (int i=0; i<3; i++)
				h_Xc[i] = pre_Xc[i];
		}
		if (h_Ycx != h_Ycx || h_Ycy != h_Ycy || h_Ycz != h_Ycz) {
			for (int i=0; i<3; i++)
				h_Yc[i] = pre_Yc[i];
		}
		////////////////

		findRTfromS(h_Xc, h_Yc, h_S, h_R, h_t, error);
		if (*error)
			break;

		//STOP_TIMER(timerAfterSVD);

		///////////////////////////////////////////////////////////////////////////////////// 
		// copy R,t to device

		//START_TIMER(timerRT);

		

		//STOP_TIMER(timerRT);

		///////////////////////////////////////////////////////////////////////////////////// 

#ifndef NOVIEWER
		if(!param.noviewer)
			UpdatePointCloud2(Ysize, param.points2, h_Y, h_R, h_t);
#endif


		sigma_p2 *= sigma_factor;
	}


	/////////////////////////////////////////////////////////////////////////////////////////
	/////////////////////////////////////////////////////////////////////////////////////////

	//CUDA_SAFE_CALL( hipDeviceSynchronize() );
	//CUT_SAFE_CALL(	cutStopTimer(timerTotal));

	//fprintf(stderr, "comping time: %.10f [s]\n", cutGetTimerValue(timerTotal) / 1000.0f);

	if(!param.notimer){
		//fprintf(stderr, "comping time: %.10f [s]\n", cutGetTimerValue(timerTotal) / 1000.0f);
		///fprintf(stderr, "Average %.10f [s] for %s\n", cutGetAverageTimerValue(timerUpdateA)  / 1000.0f, "updateA");
		//fprintf(stderr, "Average %.10f [s] for %s\n", cutGetAverageTimerValue(timerAfterSVD) / 1000.0f, "afterSVD");
		//fprintf(stderr, "Average %.10f [s] for %s\n", cutGetAverageTimerValue(timerRT) / 1000.0f, "RT");

		//CUT_SAFE_CALL(cutDeleteTimer(timerTotal));
		//CUT_SAFE_CALL(cutDeleteTimer(timerUpdateA));
		//CUT_SAFE_CALL(cutDeleteTimer(timerAfterSVD));
		//CUT_SAFE_CALL(cutDeleteTimer(timerRT));
	}

}
/***************************/
