#include "hip/hip_runtime.h"
//-----------------------------------------------------------------------------------------
//	This file is part of the "USC IRIS 3D face modeler" 
//      developed at the University of Southern California by
//      Matthias Hernandez, Jongmoo Choi, Gerard Medioni, 
//      Published: Laser Scan Quality 3-D Face Modeling Using a Low-Cost Depth Camera, EUSIPCO 2012.
//-----------------------------------------------------------------------------------------
//      Copyright (c) 2012 University of Southern California.  All Rights Reserved.


//#include "stdafx.h"

#include "Modeling_cu.h"
#include <>

float overlap_history[5]={THRESHOLD_OVERLAP, THRESHOLD_OVERLAP, THRESHOLD_OVERLAP, THRESHOLD_OVERLAP, THRESHOLD_OVERLAP};
float threshold_overlap=0.0f;

/*************************************************************/
/* Apply the inverse of the transformation (R,t) to the face */
//__global__  
__device__  void d_applyInvTransformation(float *face, float *R, float *t, int nb) {
	//int i = threadIdx.x;
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	if (x<nb) {
		float X = face[x]		- t[0];
		float Y = face[x+MAX_I] - t[1];
		float Z = face[x+MAX_I2] - t[2];

		face[x]			= (R[0]*X + R[3]*Y + R[6]*Z );
		face[x+MAX_I]	= (R[1]*X + R[4]*Y + R[7]*Z );
		face[x+MAX_I2]	= (R[2]*X + R[5]*Y + R[8]*Z ) - CYLINDER_Z;

		//out[x]			= (R[0]*X + R[3]*Y + R[6]*Z );
		//out[x+MAX_I]	= (R[1]*X + R[4]*Y + R[7]*Z );
		//out[x+MAX_I2]	= (R[2]*X + R[5]*Y + R[8]*Z );
	}
}
/*******************************/




/*******************************************/
/* Convert the face to a cylindrical model */
// face_C contains the ro,theta values
// face_I contains the indexes
//__global__  
__device__  void d_convert2Cylindrical(float *face, float *face_C, int *face_I, float *dtdy, int nb) {
	int ix = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (ix<nb) {
		int iy = ix + MAX_I;
		int iz = iy + MAX_I;
		
		float x = face[ix];
		float y = face[iy];
		float z = face[iz];
		
		// Compute cylindrical Coordinate
		face_C[ix] = pow( (x*x+z*z), 0.5f);		
		if (x<0.0f)
			face_C[iy] = (atan(z/x)+PI)*RAD2DEG;
		if (x>0.0f) {
			if (z>=0.0f)
				face_C[iy] = atan(z/x)*RAD2DEG;
			else
				face_C[iy] = (atan(z/x)+_2PI)*RAD2DEG;
		}
		if (x==0.0f) {
			if (z>0.0f)
				face_C[iy] = 90.0f;
			if (z<0.0f)
				face_C[iy] = 270.0f;
			if (z==0.0f)
				face_C[iy] = 180.0f;
		}
		float	i_tf = ((float)THETA_EXPAND*(face_C[iy])-(float)SHIFT_T+0.5f),
				i_yf = (Y_EXPAND*(-y+1.0f)+0.5f);
		// Find the row/column on the image
		int i_tc = (int)i_tf;
		int i_yc = (int)i_yf;

		// Set the weights
		dtdy[ix]	= fabs(i_tf-(float)i_tc);
		dtdy[iy]	= fabs(i_yf-(float)i_yc);

		dtdy[ix] = (dtdy[ix]>=1.0f ? 1.0f : dtdy[ix]);
		dtdy[iy] = (dtdy[iy]>=1.0f ? 1.0f : dtdy[iy]);

		// Shift to put the nose in the middle
		if (face_C[iy]<=SHIFT_T) 
			i_tc += THETA_MAX;	

		// Find the index
		face_I[ix]	=	i_yc*THETA_MAX + i_tc;	

		//if (i_yc>150 && fabs(face_C[ix])>0.0f)
		//	face_C[ix] = 0.0f;
	}

}
/*******************************/






/**********************************************/
/* Compute the overlapping error for pixel ix */
// Overlap contains the error
// Defined is used to compute the number of defined points
//__global__  
__device__ void d_computeOverlap(float *face_C, int *face_I, float *dtdy, float *model, float *overlap, bool *defined, int nb) {
	int ix = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (ix<nb) {
		int im = face_I[ix];

		if (model[im] != 0.0f) {
			overlap[ix] = pow( (face_C[ix]-model[im])*(face_C[ix]-model[im]), 0.5f);//fabs(face_C[ix]-model[im]);
			defined[ix] = true;
		} else {
			overlap[ix] = 0.0f;
			defined[ix] = false;
		}

		if (fabs(overlap[ix]) > 100.0f)
			overlap[ix]=0.0f;
	}
	
	
	/*int nb_d=0;

	if (ix<nb) {
		overlap[ix] = 0.0f;
		
		float	dt = fabs(dtdy[ix]),
				dy = fabs(dtdy[ix+MAX_I]);

		float	w0 = (1.0f-dt)*(1.0f-dy),
				w1 = (dt)*(1.0f-dy),
				w2 = (1.0f-dt)*(dy),
				w3 = (dt)*(dy);

		int im = face_I[ix];
		
		int im1=im+1,
			im2=im+THETA_MAX,
			im3=im2+1;

		if (im<MAX_I){
			if (model[im] != 0.0f && w0>MIN_WEIGHT) {
				overlap[ix] += w0*pow( (face_C[ix]-model[im])*(face_C[ix]-model[im]), 0.5f);
				nb_d++;
			} 
		}
		if (im1<MAX_I){
			if (model[im1] != 0.0f && w1>MIN_WEIGHT) {
				overlap[ix] += w1*pow( (face_C[ix]-model[im1])*(face_C[ix]-model[im1]), 0.5f);
				nb_d++;
			} 
		}
		if (im2<MAX_I){
			if (model[im2] != 0.0f && w2>MIN_WEIGHT) {
				overlap[ix] += w2*pow( (face_C[ix]-model[im2])*(face_C[ix]-model[im2]), 0.5f);
				nb_d++;
			} 
		}
		if (im3<MAX_I){
			if (model[im3] != 0.0f && w3>MIN_WEIGHT) {
				overlap[ix] += w3*pow( (face_C[ix]-model[im3])*(face_C[ix]-model[im3]), 0.5f);				
				nb_d++;
			} 
		}

		if (nb_d>0) {
			overlap[ix] /= (float)nb_d;
			defined[ix] = true;
		} else {
			overlap[ix] = 0.0f;
			defined[ix] = false;
		}

		if (fabs(overlap[ix]) > 100.0f)
			overlap[ix]=0.0f;
	}*/
}
/*******************************/





/********************/
/* Update the model */
__global__ void d_preparePoints(float *face, float *R, float *t, float *face_C, int *face_I, float *dtdy, float *model, float *overlap, bool *defined, int nb) {
	d_applyInvTransformation(face, R, t, nb);
	d_convert2Cylindrical(face, face_C, face_I, dtdy, nb);
	d_computeOverlap(face_C, face_I, dtdy, model, overlap, defined, nb);
}
/*******************************/




/********************/
/* Update the model */
__global__ void d_updateModel(float *face_C, int *face_I, float *dtdy, float *overlap, float *model, float *meann, int nb) {
	int ix = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (abs(ix) < nb) {
		int	im0 = face_I[ix],
			im1 = im0+1,
			im2 = im0+THETA_MAX,
			im3 = im2+1;

		float	dt = fabs(dtdy[ix]);
		float	dy = fabs(dtdy[ix+MAX_I]);
		float	w0 = fabs((1.0f-dt)*(1.0f-dy));
		float	w1 = fabs((dt)*(1.0f-dy));
		float	w2 = fabs((1.0f-dt)*(dy));
		float	w3 = fabs((dt)*(dy));

		float v=fabs(face_C[ix]);

		float value=0.0f;

		if (fabs(overlap[ix])<THRESHOLD_UPDATE) {
			if (v>0.0f) {
				if (im0 < MAX_IMG_INDEX_1 && w0>MIN_WEIGHT){
					value = (((meann[im0])*(model[im0]) + (w0)*(v)) / ((meann[im0])+(w0)));

					model[im0] = value;
					meann[im0]+=w0;
				}
				if (im1 < MAX_IMG_INDEX_1 && w1>MIN_WEIGHT) {
					value = (((meann[im1])*(model[im1]) + (w1)*(v)) / ((meann[im1])+(w1)));

					model[im1] = value;
					meann[im1]+=w1;
				}
				if (im2 < MAX_IMG_INDEX_1 && w2>MIN_WEIGHT) {	
					value = (((meann[im2])*(model[im2]) + (w2)*(v)) / ((meann[im2])+(w2)));

					model[im2] = value;
					meann[im2]+=w2;
				}
				if (im3 < MAX_IMG_INDEX_1 && w3>MIN_WEIGHT) { 
					value = (((meann[im3])*(model[im3]) + (w3)*(v)) / ((meann[im3])+(w3)));

					model[im3] = value;
					meann[im3]+=w3;
				}
			}
		}


/*
		if (model[im0]>RO_MAX) {
			model[im0] = 0.0f;//face_C[ix];
			meann[im0] = 0.0f;//w0;
		}
		if (model[im1]>RO_MAX) {
			model[im1] = 0.0f;//face_C[ix];
			meann[im1] = 0.0f;//w1;
		}
		if (model[im2]>RO_MAX) {
			model[im2] = 0.0f;//face_C[ix];
			meann[im2] = 0.0f;//w2;
		}
		if (model[im3]>RO_MAX) {
			model[im3] = 0.0f;//face_C[ix];
			meann[im3] = 0.0f;//w3;
		}*/

	}
	
}
/*******************************/

/******************************/
/* Update the unwrapped image */
__global__ void d_updateUnwrappedImage(float *model, char *dData, int w, int h) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < w && y < h) {
		int i1 = y*w+x,
			i3 = 3*i1;

		dData[i3] = ((model[i1]<1.0f) ? (char)(model[i1]*255.0f + 0.5f)  : 255);
		dData[i3+1] = dData[i3];
		dData[i3+2] = dData[i3];
	}
}
/*******************************/





/******************************/
/* Update the unwrapped image */
__global__ void d_convertToXYZRGB(float *model, float *modelXYZ, float *modelRGB, char *iData, bool *defined_C, float cx, float cy, float cz, float normalize, int w, int h) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

	float ro=0.0f, theta=0.0f;
	float xx=0.0f, yy=0.0f, zz=0.0f;
	float xR=0.0f, yR=0.0f, zR=0.0f;

	float ixf=0.0f, iyf=0.0f;
	int ixi=0, iyi=0;
	float dx=0.0f, dy=0.0f;
	float w00=0.0f, w01=0.0f, w10=0.0f, w11=0.0f;

	float R=0,G=0,B=0;

	int iii00=0, iii01=0, iii10=0, iii11=0;
	float b00=0, b01=0, b10=0, b11=0;
	float g00=0, g01=0, g10=0, g11=0;
	float r00=0, r01=0, r10=0, r11=0;
	int col=0;


    if (x < w && y < h) {
		int i = y*w+x,
			i1 = i+MAX_IMG_INDEX_1,
			i2 = i+MAX_IMG_INDEX_2,
			i3 = 3*i;

		if (defined_C[i]) {
			// Find the cylindric coordinate ro/theta
			ro = model[i];
			col = (i%THETA_MAX)/(THETA_MAX/360);
			
			if (theta<(THETA_MAX-SHIFT_T))
				theta = (float)(col+SHIFT_T) * DEG2RAD;
			else
				theta = (float)(col+SHIFT_T-THETA_MAX) * DEG2RAD;
			
			// Find the cartesian coordinate
			yy = -(((float)(i-col)/(float)THETA_MAX)/(Y_EXPAND)-1.0f);
			xx = ro * cos(theta);
			zz = ro * sin(theta);

			// Come back to real coordinate 
			xR = (xx/1.05f-0.02f)*normalize+cx;
			yR = (yy+0.02f)*normalize+cy;

			//xR = (xx)*normalize+cx;
			//yR = (yy)*normalize+cy;
			zR = (zz+CYLINDER_Z)*normalize+cz;	//TODO: CALIBRATE TO FIX THIS

			//#define CALIB_T0	0.0f//3.0f
			//#define CALIB_T1	0.0f//3.0f
			//#define CALIB_T2	0.0f
			//#define ANGLE_X		-0.00f//-0.005f
			//#define ANGLE_Y		0.00f

			/*#define ANGLE_X		0.021000f
			#define ANGLE_Y		-0.00280f//-0.001200f
			#define CALIB_T0	3.85000f//3.785000f
			#define CALIB_T1	-3.88f//-5.88f
			#define CALIB_T2	0.0f

			#define cY			cos(ANGLE_Y)
			#define cX			cos(ANGLE_X)
			#define sY			sin(ANGLE_Y)
			#define sX			sin(ANGLE_X)
			#define CALIB_R0	cY
			#define CALIB_R1	sX*sY
			#define CALIB_R2	cX*sY
			#define CALIB_R3	0.0f
			#define CALIB_R4	cX
			#define CALIB_R5	-sX
			#define CALIB_R6	-sY
			#define CALIB_R7	sX*cY
			#define CALIB_R8	cX*cY

			xR = CALIB_R0*xR+CALIB_R1*yR+CALIB_R2*zR+CALIB_T0;
			yR = CALIB_R3*xR+CALIB_R4*yR+CALIB_R5*zR+CALIB_T1;
			zR = CALIB_R6*xR+CALIB_R7*yR+CALIB_R8*zR+CALIB_T2;*/

			//if (IS_HR)
			yR+=2.0f;

			// Project onto the image plane
			ixf = (xR/zR/XtoZ + 0.5f)*(float)XN_HR_X_RES;
			iyf = (-yR/zR/YtoZ + 0.5f)*(float)XN_HR_Y_RES;

			// Take the integer part and decimal part
			ixi = (int)ixf; 
			iyi = (int)iyf;
			dx = ixf-(float)ixi;
			dy = iyf-(float)iyi;
			//ixi *= 3;
			//iyi *= 3;
			
			// Find the corresponding indexes onto the image
			iii00	= (iyi*XN_HR_X_RES	+ ixi)*3;
			iii10	= iii00 + 3;
			iii01	= iii00 + XN_HR_X_RES3;
			iii11	= iii01 + 3;

			if (iii11<MAX_I_HR3 && iii00>=0) {
				// Find the weights
				w00 = (1-dx) * (1-dy);
				w10 = dx * (1-dy);
				w01 = (1-dx) * dy;
				w11 = dx * dy;

				b00 = (iData[iii00]<0 ? 255.0f-fabs((float)iData[iii00]): (float)iData[iii00]);
				b10 = (iData[iii10]<0 ? 255.0f-fabs((float)iData[iii10]): (float)iData[iii10]);
				b01 = (iData[iii01]<0 ? 255.0f-fabs((float)iData[iii01]): (float)iData[iii01]);
				b11 = (iData[iii11]<0 ? 255.0f-fabs((float)iData[iii11]): (float)iData[iii11]);

				g00 = (iData[iii00+1]<0 ? 255.0f-fabs((float)iData[iii00+1]): (float)iData[iii00+1]);
				g10 = (iData[iii10+1]<0 ? 255.0f-fabs((float)iData[iii10+1]): (float)iData[iii10+1]);
				g01 = (iData[iii01+1]<0 ? 255.0f-fabs((float)iData[iii01+1]): (float)iData[iii01+1]);
				g11 = (iData[iii11+1]<0 ? 255.0f-fabs((float)iData[iii11+1]): (float)iData[iii11+1]);
				
				r00 = (iData[iii00+2]<0 ? 255.0f-fabs((float)iData[iii00+2]): (float)iData[iii00+2]);
				r10 = (iData[iii10+2]<0 ? 255.0f-fabs((float)iData[iii10+2]): (float)iData[iii10+2]);
				r01 = (iData[iii01+2]<0 ? 255.0f-fabs((float)iData[iii01+2]): (float)iData[iii01+2]);
				r11 = (iData[iii11+2]<0 ? 255.0f-fabs((float)iData[iii11+2]): (float)iData[iii11+2]);

				// Find R,G,B	-values should be between 0 and 1-		
				B = (w00*b00	+ w10*b10 
								+ w01*b01	
								+ w11*b11)	/ 255.0f;
				G = (w00*g00	+ w10*g10
								+ w01*g01 
								+ w11*g11)	/ 255.0f;
				R = (w00*r00	+ w10*r10
								+ w01*r01
								+ w11*r11)	/ 255.0f;
			}

			// Save into the file
			modelXYZ[i]  = -xx;
			modelXYZ[i1] =  yy;
			modelXYZ[i2] = -zz;

			modelRGB[i3]   = R;
			modelRGB[i3+1] = G;
			modelRGB[i3+2] = B;
		}
	}
}
/*******************************/



/*********************************/
/* Entire model updating process */
void convertToXYZRGB(	float *h_model, float *d_model, 
						float *h_modelXYZ, float *d_modelXYZ, 
						float *h_modelRGB, float *d_modelRGB,
						char *h_iData, char *d_iData,
						bool *h_defined_C, bool *d_defined_C,
						Voxel c, float normalize) {
	
	dim3 gridSize((THETA_MAX + 16 - 1) / 16, (Y_MAX + 16 - 1) / 16);
	dim3 blockSize(16, 16);

	copyHostToCUDA(model, MAX_IMG_INDEX_1);
	copyHostToCUDA_char(iData, MAX_I3);	// To modify if want HR
	copyHostToCUDA_bool(defined_C, MAX_IMG_INDEX_1);

	d_convertToXYZRGB<<< gridSize, blockSize >>>(d_model, d_modelXYZ, d_modelRGB, d_iData, d_defined_C, c.getXr(), c.getYr(), c.getZr(), normalize, THETA_MAX, Y_MAX);	
	
	copyCUDAToHost(modelXYZ, MAX_IMG_INDEX_3);
	copyCUDAToHost(modelRGB, MAX_IMG_INDEX_3);
}
/*******************************/










/*********************************/
/* Entire model updating process */
void updateModel(	float *h_face, float *d_face,
					int nb,
					float *h_R, float *d_R, 
					float *h_t, float *d_t,
					float *d_face_C,
					int *d_face_I,
					float *d_dtdy,
					float *h_model, float *d_model,
					float *h_overlap, float *d_overlap,
					bool *h_defined, bool *d_defined,
					char *h_dData, char *d_dData,
					float *d_meann,
					bool *wrongRegistration,
					int NB_FRAMES) {
	
	dim3 gridSize((THETA_MAX + BLOCK_SIZE - 1) / 16, (Y_MAX + BLOCK_SIZE - 1) / BLOCK_SIZE);
	dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
	int nb_g = (nb + BLOCK_SIZE-1) / BLOCK_SIZE;

	copyHostToCUDA(t, 3);
	copyHostToCUDA(R, 9);
	//copyHostToCUDA(face, MAX_I3);
	copyHostToCUDA_XYZ(face, nb);

	//synchronizeCUDA();
	//d_preparePoints<<< nb, 1 >>>(d_face, d_R, d_t, d_face_C, d_face_I, d_dtdy, d_model, d_overlap, d_defined, nb);	
	d_preparePoints<<< nb_g, BLOCK_SIZE >>>(d_face, d_R, d_t, d_face_C, d_face_I, d_dtdy, d_model, d_overlap, d_defined, nb);	
	
	//copyCUDAToHost(overlap, MAX_I);
	//copyCUDAToHost_bool(defined, MAX_I);

	int		nb_overlap=0;
	float	overlap=0.0f,
			av_overlap=0.0f;
	//////////////////////////////////////////////////////////////////////////
	// VERSION 2
	copyCUDAToHost(overlap, MAX_I);
	copyCUDAToHost_bool(defined, MAX_I);
	for (int i=0; i<nb; i++){
		if (h_defined[i]) {
			nb_overlap++;
			av_overlap += h_overlap[i];
		}
	}
	
	
	
	
	
	
	//////////////////////////////////////////////////////////////////////////
	// VERSION 1
	//av_overlap = hipblasSasum(nb, d_overlap, 1);
	//nb_overlap = nb-(int)hipblasSasum(nb, (float *)d_defined, 1);

	if (nb_overlap > 0)
		av_overlap /= (float)nb_overlap;

	// Compute the average of the values greater than the average
	if (av_overlap > 0.0f) {
		//////////////////////////////////////////////////////////////////////////
		// VERSION 1
		// copyCUDAToHost(overlap, MAX_I);
		// copyCUDAToHost_bool(defined, MAX_I);
		nb_overlap=0;
		overlap=0.0f;
		for (int i=0; i<nb; i++){
			if (h_defined[i] && h_overlap[i]>av_overlap) {
				nb_overlap++;
				overlap += h_overlap[i];
			}
		}
		if (nb_overlap > 0)
			overlap /= (float)nb_overlap;
	}

	// Set the adaptive threshold
	if (NB_FRAMES > 0 && NB_FRAMES <= 6) 
		overlap_history[NB_FRAMES-1] = overlap;
	if (NB_FRAMES==0)
		threshold_overlap = THRESHOLD_OVERLAP;
	if (NB_FRAMES==6) 
		threshold_overlap = (overlap_history[0]+overlap_history[1]+overlap_history[2]+overlap_history[3]+overlap_history[4])/2.0f;

	//printf("%f // %f\n", overlap, threshold_overlap);



	// Detect wrong registrations
	if (overlap < threshold_overlap || overlap != overlap) {
		//synchronizeCUDA();
		d_updateModel<<< nb_g, BLOCK_SIZE >>>(d_face_C, d_face_I, d_dtdy, d_overlap, d_model, d_meann, nb);	

		//d_updateModel<<< nb, 1 >>>(d_face_C, d_face_I, d_dtdy, d_overlap, d_model, d_meann, nb);	
		(*wrongRegistration) = false;
	} else
		(*wrongRegistration) = true;

	
	////synchronizeCUDA();
	d_updateUnwrappedImage<<< gridSize, blockSize>>>(d_model, d_dData, THETA_MAX, Y_MAX ); 

	copyCUDAToHost_char(dData, MAX_IMG_INDEX_3);

	
	
	
	/*copyCUDAToHost(model, MAX_IMG_INDEX_1);
	float *h_meann = new float[MAX_IMG_INDEX_1];
	float *h_face_C = new float[MAX_IMG_INDEX_1];
	copyCUDAToHost(meann, MAX_IMG_INDEX_1);
	copyCUDAToHost(face_C, MAX_IMG_INDEX_1);
	for (int i=0;i<MAX_IMG_INDEX_1; i++) {
		if (h_face_C[i]>RO_MAX) {
			printf("face %f\n", h_face_C[i]);			
		}
		
		if (h_model[i]>RO_MAX) {
			printf("%f %f\n", h_model[i], h_meann[i]);			
		}
	}



	float *h_dtdy = new float[MAX_I2];
	copyCUDAToHost(dtdy, MAX_I2);
	//for (int i=0; i<nb; i++)
	//	printf("%f %f\n", h_dtdy[i], h_dtdy[i+MAX_I]);
	delete[] h_dtdy;
	delete[] h_meann;
	delete[] h_face_C;*/

}
/*******************************/




